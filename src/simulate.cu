#include "hip/hip_runtime.h"

#include <iostream>
#include <set>
#include <string>
#include <chrono>
#include <thread>
#include "simulate.cuh"
#include <algorithm>
#include <limits>
#include <cstddef>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <boost/serialization/string.hpp>
#include <boost/serialization/vector.hpp>
#include <boost/serialization/map.hpp>
#include <boost/serialization/deque.hpp>
//#include <boost/serialization/unordered_map.hpp>
#include <boost/serialization/access.hpp>
#include <boost/archive/text_iarchive.hpp>
#include <boost/archive/text_oarchive.hpp>
#define COMPRESS_BASE 65536
#define COMPRESS_WIDTH 8
#define UNKNOWN 2
#define STATE_NUM 8
#define MAX_IN_NUM 6
#define MAX_OUT_NUM 2
#define UPDATE_N_THREADS_PER_BLOCK 256
#define N_THREADS_PER_BLOCK 512
#define _N_THREADS_PER_BLOCK 1024
#define N_TIMES_PER_THREAD 32
#define MIN_THREAD_NUM 4
#define MAX_BLOCKS 65535
#define _MAX_BLOCKS 65535
//#define SIM_START 10000
//#define SIM_START 0
//#define SIM_END 100000001 
//#define SIM_END 20000001 
//#define SIM_END 2539945010
//#define SIM_END 2972036001
#define MAX_NUM 100000001
typedef struct {
    int x;
    int y;
} duo;
class inter
{
public:
    inter(){}
    ~inter(){}
    Instance* find_inst(int id){
        return p.find_inst(id);
    }

    /* data */
    PreProcess p;
    std::vector<std::vector<int> > levels;
    std::map<std::string, int> initial_net_map;

    std::vector<unsigned int> data_in_num_start;
    std::vector<unsigned int> data_out_num_start;
    std::vector<unsigned int> delay_start;
    std::vector<unsigned int> delay_width;
    std::vector<unsigned int> functions_start;
    std::vector<unsigned int> functions_width;

    std::vector<short> data_in_num;
    std::vector<short> data_out_num;
    std::vector<unsigned int> val_num_start;
    std::vector<short> delay_val_num;
    std::vector<short> functions_func_num;
    std::vector<short> functions_val_num;
    std::vector<short> host_delay_edges;
    std::vector<short> host_in_bit;
    std::vector<short> host_out_bit;
    std::vector<float> host_rise_val;
    std::vector<float> host_fall_val;
    std::vector<std::map<std::string, std::vector<unsigned int> > > OutMaps;

    std::vector<short> host_functions;
    template<class Archive>
    void serialize(Archive & ar, const unsigned int version) //
    {            //ar.register_type(static_cast<ModuleType *>(NULL));
        ar & p;
        ar & levels;
        ar & initial_net_map;
        ar & data_in_num_start;
        ar & data_out_num_start;
        ar & delay_start;
        ar & delay_width;
        ar & functions_start;
        ar & functions_width;
        ar & data_in_num;
        ar & data_out_num;
        ar & val_num_start;
        ar & delay_val_num;
        ar & functions_func_num;
        ar & functions_val_num;
        ar & host_delay_edges;
        ar & host_in_bit;
        ar & host_out_bit;
        ar & host_rise_val;
        ar & host_fall_val;
        ar & host_functions;
        ar & OutMaps;
    }
};
using namespace std;
std::unordered_map<std::string, std::vector<unsigned long long> > Result;
std::unordered_map<std::string, std::vector<unsigned long long> > OtherResult;
std::unordered_set<std::string> pin_bits_set;
__device__ unsigned int dev_out_width;

void DumpSaif(ofstream &outf){
    for(auto &it:Result){
        string net_name = it.first;
        if(net_name.find("[") != std::string::npos){
            pin_bits_set.erase(net_name);
        }
        std::vector<unsigned long long> v = it.second;
        outf << "      (" << net_name << endl;
        outf << "         (T0 " << v[0] << ") (T1 " << v[1] << ") (TX " << v[2] << ")" << endl;
        outf << "      )" << endl;
    }
}

void DumpSaif(ofstream &outf, std::unordered_map<std::string, std::vector<unsigned long long> >& inResult){
    for(auto &it:inResult){
        string net_name = it.first;
        if(net_name.find("[") != std::string::npos){
            pin_bits_set.erase(net_name);
        }
        std::vector<unsigned long long> v = it.second;
        outf << "      (" << net_name << endl;
        outf << "         (T0 " << v[0] << ") (T1 " << v[1] << ") (TX " << v[2] << ")" << endl;
        outf << "      )" << endl;
    }
}

void DumpSaif(ofstream &outf, std::map<std::string, std::vector<unsigned long long> >& pinResult, VCDTime start, VCDTime end){
    for(auto &it:pinResult){
        string net_name = it.first;
        if(net_name.find("[") != std::string::npos){
            pin_bits_set.erase(net_name);
        }
        std::vector<unsigned long long> v = it.second;
        outf << "      (" << net_name << endl;
        outf << "         (T0 " << (v[0]?(end - start):0) << ") (T1 " << (v[1]?(end - start):0) << ") (TX " << 0 << ")" << endl;
        outf << "      )" << endl;
    }
}

void DumpSaif(ofstream &outf, std::vector<std::map<std::string, std::vector<unsigned int> > > &OutMaps,
    std::vector<unsigned int*> &out_times,std::vector<short*> &out_values,std::vector<unsigned int*> &out_sizes, std::vector<unsigned int*> &out_starts,
    VCDTime start, VCDTime end){
    unsigned size = OutMaps.size();
    for(unsigned i = 0; i < size; ++i){
        for(auto &it:OutMaps[i]){
            string net_name = it.first;
            if(net_name.find("[") != std::string::npos){
                pin_bits_set.erase(net_name);
            }
            outf << "      (" << net_name << endl;
            std::vector<unsigned int> v = it.second;
            unsigned int cur_level = i;
            unsigned int cur_pos = v[0];
            unsigned int cur_out_order = v[1];
            unsigned int cur_size = *(out_sizes[cur_level] + cur_pos * MAX_OUT_NUM + cur_out_order);
            unsigned int tmp_step = *(out_sizes[cur_level] + cur_pos * MAX_OUT_NUM);
            unsigned int cur_start = *(out_starts[cur_level] + cur_pos) + cur_out_order * tmp_step; 

            unsigned int *cur_time = out_times[cur_level] + cur_start;
            short *cur_value = out_values[cur_level] + cur_start / STATE_NUM;
            short offset = cur_start % STATE_NUM;
            unsigned long long time_1=0;
            unsigned long long time_0=0;
            unsigned long long time_x=0;
            unsigned long long time_z=0;
            unsigned long long last_time = start;
            VCDBit last_state = VCD_X;
            // find real start
            unsigned pos_val_start = 0;
            short start_data = cur_value[pos_val_start];
            short data;
            short shift = (3 << 14);
            int o;
            for(o = 0; o < offset; ++o){
                start_data = (start_data << 2);
            }
            //
            for (unsigned l = 0; l < cur_size; ++l)
            {
                short data = ((start_data & shift) >> 14) & 3;
                ++o;
                start_data = (start_data << 2);
                if(o == STATE_NUM){
                    ++pos_val_start;
                    o = 0;
                    start_data = cur_value[pos_val_start];
                }
                VCDTime cur_t = cur_time[l];
                VCDBit cur_val = VCDBit(data);
                if (cur_t <= start)
                {
                    last_state = cur_val;
                    continue;
                }
                if (cur_t > end)
                {
                    break;
                }
                if (last_state == VCD_X)
                {
                    time_x += cur_t - last_time;
                }
                else if (last_state == VCD_0)
                {
                    time_0 += cur_t - last_time;
                }
                else if (last_state == VCD_1)
                {
                    time_1 += cur_t - last_time;
                }
                else if (last_state == VCD_Z)
                {
                    time_z += cur_t - last_time;
                }
                last_time = cur_t;
                last_state = cur_val;
            }
            if (last_time < end)
            {
                if (last_state == VCD_X)
                {
                    time_x += end - last_time;
                }
                else if (last_state == VCD_0)
                {
                    time_0 += end - last_time;
                }
                else if (last_state == VCD_1)
                {
                    time_1 += end - last_time;
                }
                else if (last_state == VCD_Z)
                {
                    time_z += end - last_time;
                }
            }   

            outf << "         (T0 " << time_0 << ") (T1 " << time_1 << ") (TX " << time_x+time_z << ")" << endl;
            outf << "      )" << endl;
            std::vector<unsigned long long> temp = {time_0, time_1, time_x + time_z};
            OtherResult[net_name] = temp;
        }
    }
}

void DumpSaif(std::ofstream &outf, std::unordered_set<std::string> &InMaps_bus, std::unordered_set<std::string> &InMaps_bit, 
    std::unordered_map<std::string, std::vector<unsigned int> > &times, std::unordered_map<std::string, std::vector<short> > &values, 
    std::unordered_map<std::string, unsigned int> &sizes,
    std::vector<std::vector<unsigned int> > &vec_times, std::vector<std::vector<short> > &vec_values, std::vector<unsigned int> &vec_sizes,
    std::unordered_map<std::string, std::string> &hash_name_pair, std::unordered_map<std::string, unsigned int> &hash_index_pair,
    VCDTime start, VCDTime end)
{
    for(auto &it:InMaps_bus){
        if(it.find("[") != std::string::npos){
            pin_bits_set.erase(it);
        }
        unsigned cur_size = sizes[it];
        unsigned int *pos = &(*(times[it].begin()));
        short *pos_val = &(*(values[it].begin()));
        outf << "      (" << it << endl;
        unsigned long long time_1=0;
        unsigned long long time_0=0;
        unsigned long long time_x=0;
        unsigned long long time_z=0;
        unsigned long long last_time = start;
        VCDBit last_state = VCD_X;
        //
        for (unsigned l = 0; l < cur_size; ++l)
        {
            VCDTime cur_t = pos[l];
            VCDBit cur_val = VCDBit(pos_val[l]);
            if (cur_t <= start)
            {
                last_state = cur_val;
                continue;
            }
            if (cur_t > end)
            {
                break;
            }
            if (last_state == VCD_X)
            {
                time_x += cur_t - last_time;
            }
            else if (last_state == VCD_0)
            {
                time_0 += cur_t - last_time;
            }
            else if (last_state == VCD_1)
            {
                time_1 += cur_t - last_time;
            }
            else if (last_state == VCD_Z)
            {
                time_z += cur_t - last_time;
            }
            last_time = cur_t;
            last_state = cur_val;
        }
        if (last_time < end)
        {
            if (last_state == VCD_X)
            {
                time_x += end - last_time;
            }
            else if (last_state == VCD_0)
            {
                time_0 += end - last_time;
            }
            else if (last_state == VCD_1)
            {
                time_1 += end - last_time;
            }
            else if (last_state == VCD_Z)
            {
                time_z += end - last_time;
            }
        }   

        outf << "         (T0 " << time_0 << ") (T1 " << time_1 << ") (TX " << time_x+time_z << ")" << endl;
        outf << "      )" << endl;
        std::vector<unsigned long long> temp = {time_0, time_1, time_x + time_z};
        OtherResult[it] = temp;
    }
    for(auto &it:InMaps_bit){
        if(it.find("[") != std::string::npos){
            pin_bits_set.erase(it);
        }
        string hash = hash_name_pair[it];
        unsigned _idx = hash_index_pair[hash];
        unsigned cur_size = vec_times[_idx].size();//vec_sizes[_idx];//
        unsigned int *pos = &(*(vec_times[_idx].begin()));
        short *pos_val = &(*(vec_values[_idx].begin()));
        outf << "      (" << it << endl;
        unsigned long long time_1=0;
        unsigned long long time_0=0;
        unsigned long long time_x=0;
        unsigned long long time_z=0;
        unsigned long long last_time = start;
        VCDBit last_state = VCD_X;
        //
        for (unsigned l = 0; l < cur_size; ++l)
        {
            VCDTime cur_t = pos[l];
            VCDBit cur_val = VCDBit(pos_val[l]);
            if (cur_t <= start)
            {
                last_state = cur_val;
                continue;
            }
            if (cur_t > end)
            {
                break;
            }
            if (last_state == VCD_X)
            {
                time_x += cur_t - last_time;
            }
            else if (last_state == VCD_0)
            {
                time_0 += cur_t - last_time;
            }
            else if (last_state == VCD_1)
            {
                time_1 += cur_t - last_time;
            }
            else if (last_state == VCD_Z)
            {
                time_z += cur_t - last_time;
            }
            last_time = cur_t;
            last_state = cur_val;
        }
        if (last_time < end)
        {
            if (last_state == VCD_X)
            {
                time_x += end - last_time;
            }
            else if (last_state == VCD_0)
            {
                time_0 += end - last_time;
            }
            else if (last_state == VCD_1)
            {
                time_1 += end - last_time;
            }
            else if (last_state == VCD_Z)
            {
                time_z += end - last_time;
            }
        }   

        outf << "         (T0 " << time_0 << ") (T1 " << time_1 << ") (TX " << time_x+time_z << ")" << endl;
        outf << "      )" << endl;
        std::vector<unsigned long long> temp = {time_0, time_1, time_x + time_z};
        OtherResult[it] = temp;
    }
}

void DumpSaif(ofstream &outf, std::map<std::string, std::string> &assign_pairs, 
    //std::unordered_map<std::string, std::vector<unsigned long long> >& inResult, 
    std::map<std::string, std::vector<unsigned long long> >& pinResult,
    VCDTime start, VCDTime end){
    for(auto &it:assign_pairs){
        string net_name = it.first;
        if(Result.find(net_name) != Result.end() || OtherResult.find(net_name) != OtherResult.end() || pinResult.find(net_name) != pinResult.end()){
            continue;
        }
        if(net_name.find("[") != std::string::npos){
            pin_bits_set.erase(net_name);
        }
        string des_name = it.second;
        outf << "      (" << net_name << endl;
        if(Result.find(des_name) != Result.end()){
            std::vector<unsigned long long> v = Result[des_name];
            outf << "         (T0 " << v[0] << ") (T1 " << v[1] << ") (TX " << v[2] << ")" << endl;
        }
        else if(OtherResult.find(des_name) != OtherResult.end()){
            std::vector<unsigned long long> v = OtherResult[des_name];
            outf << "         (T0 " << v[0] << ") (T1 " << v[1] << ") (TX " << v[2] << ")" << endl;
        }
        /*else if(inResult.find(des_name) != inResult.end()){
            std::vector<unsigned long long> v = inResult[des_name];
            outf << "         (T0 " << v[0] << ") (T1 " << v[1] << ") (TX " << v[2] << ")" << endl;
        }*/
        else if(pinResult.find(des_name) != pinResult.end()){
            std::vector<unsigned long long> v = pinResult[des_name];
            outf << "         (T0 " << (v[0]?(end - start):0) << ") (T1 " << (v[1]?(end - start):0) << ") (TX " << v[2] << ")" << endl;
        }
        outf << "      )" << endl;
    }
}

void DumpSaif(ofstream &outf, VCDTime start, VCDTime end)
{
    for(auto &it:pin_bits_set){
        outf << "      (" << it << endl;
        outf << "         (T0 " << 0 << ") (T1 " << 0 << ") (TX " << end - start << ")" << endl;
        outf << "      )" << endl;
    }
    outf << "   )" << endl;
}

void DumpSaif(ofstream &outf, const VCDTimeUnit time_unit, const unsigned time_res, const std::string root_name, VCDTime start, VCDTime end)
{
    unsigned int total_time = end - start;
    if (total_time <= 0)
    {
        cout << "dumpoff_time must be lower than dumpon_time" << endl;
        exit(-1);
    }
    /**/
    string tmp_tu;
    if(time_unit == TIME_S)
        tmp_tu = "s";
    else if(time_unit == TIME_MS)
        tmp_tu = "ms";
    else if(time_unit == TIME_US)
        tmp_tu = "us";
    else if(time_unit == TIME_NS)
        tmp_tu = "ns";
    else if(time_unit == TIME_PS)
        tmp_tu = "ps";
    outf << "(TIMESCALE " << time_res << " " << tmp_tu << ")" << endl;
    outf << "(DURATION " << total_time << ")" << endl;
    outf << "(INSTANCE " << root_name << endl;
    outf << "   (NET" << endl;
}



//GPU
__global__
void SimulateCuda(int cur_level,int blocks_per_inst, short *dev_datas, short *dev_out_datas, /*int *dev_in_times, int *dev_in_data, */unsigned int *dev_times, unsigned int *dev_out_times,//unsigned int *dev_data_val_num, unsigned int *dev_in_times,
                short *dev_functions, short *dev_delay_edges, short *dev_in_bit, short *dev_out_bit, float *dev_rise_val, float *dev_fall_val,
                int dev_time_unit, unsigned int *dev_total_times_start, unsigned int *dev_total_times_sizes, short *dev_data_in_num, short *dev_data_out_num, short *dev_delay_val_num,short *dev_functions_func_num, short *dev_functions_val_num, 
                unsigned int *dev_input_start, //unsigned int *dev_val_num_start,
                //unsigned int *dev_valid_width,
                short *dev_out_splited_width, unsigned int *dev_output_start, unsigned int *dev_output_size, unsigned int *dev_data_in_num_start,
                unsigned int *dev_data_out_num_start, unsigned int *dev_delay_start, unsigned int *dev_delay_width, unsigned int *dev_functions_start, unsigned int *dev_functions_width,
                unsigned int *dev_valid_width, unsigned int *dev_valid_width_start, unsigned int *dev_valid_width_size)//, int *dev_data_start)
{
    const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    const unsigned int _tid = threadIdx.x;
    
    if (_tid < N_THREADS_PER_BLOCK)
    {
        const unsigned int bid = _bid / blocks_per_inst;
        const unsigned int tid = _tid + (_bid % blocks_per_inst) * N_THREADS_PER_BLOCK;
        //const int inval_start = bid*18;
        //const int lastval_start = bid*6;
        //times flag
        const unsigned int time_width = dev_total_times_sizes[bid];
        const unsigned int time_start = dev_total_times_start[bid];
        //int time_start;        
        //__shared__ int dev_in_times[12288];
        //printf("tid:%d, time_width:%d, time_start:%d\n", tid, time_width, time_start);
        const unsigned int _start = dev_data_in_num_start[cur_level];
        //in data flag
        const short in_num = dev_data_in_num[_start+bid];
        //if(_tid == 0) printf("_start:%d\n", _start);
        //out data flag
        const short out_num = dev_data_out_num[_start+bid];
        //delay flag
        const short delay_width = dev_delay_val_num[_start+bid];
        //function flag
        const short function_num = dev_functions_func_num[_start+bid];
        //if (bid == 13 && inst_num == 1822)
        //    printf("funcnum: %hd, global_func_width: %d\n", function_num,global_func_width);
        const short width_every_function = dev_functions_val_num[_start+bid];
        //begin point
        //unsigned long long val_num_start = bid*global_val_num_width; //in datas and in times
        const unsigned int out_start = dev_output_start[bid];//bid*global_out_width;
        const unsigned int out_width_every = dev_output_size[bid];//global_out_width/out_num;
        const unsigned int valid_width_start = dev_valid_width_start[bid];//global_out_width/out_num;
        const unsigned int valid_width_size = dev_valid_width_size[bid];//global_out_width/out_num;
        const unsigned int in_start = dev_input_start[bid];
        //const int time_start = dev_time_start[bid];
        //const unsigned int val_start = dev_val_num_start[bid];

        const int func_start = dev_functions_start[cur_level]+bid*dev_functions_width[cur_level];//bid*global_func_width;
        const int func_width_every = dev_functions_width[cur_level]/function_num;
        const int delay_start = dev_delay_start[cur_level]+bid*dev_delay_width[cur_level];//bid*global_delay_width;
        //if(_tid == 0) printf("func_start:%d, %d\n", func_start,delay_start);
        // calculate
        //printf("%d start calculate\n", tid);
        short _cur_inValues[18];
        short _last_inValues[6];
        //int last_vec[6] = {2,2,2,2,2,2};
        unsigned int left, left_start, left_shift, right, mid;
        unsigned int cur_time, out_time, last_time, _last_time;
        //short cur_out, pre_out, output_data, last_value, cur, last;
        short cur_out, pre_out, output_data, last_value, cur, last;
        unsigned int i,j,k,l;
        short func_flag;
        short input_data, _input_data, input_data1;
        //__shared__ int _time_width[N_THREADS_PER_BLOCK];
        short cur_func;
        short pos, out_sense, in_sense, num_in, changed_id, edge;
        int func_in_num;
        float delay_val, factor, min_val;
        int delay_valid_id1, delay_valid_id2, delay_valid_id;  //one input driving
        unsigned int cur_start, last_start;

        short splited_time_width;
        if (time_width > N_THREADS_PER_BLOCK*N_TIMES_PER_THREAD*blocks_per_inst)
        {
            splited_time_width = time_width / (N_THREADS_PER_BLOCK*blocks_per_inst) + 1;
        }
        else
            splited_time_width = N_TIMES_PER_THREAD;
        dev_out_splited_width[bid] = splited_time_width;// + 1;
        for (i = 0; i < out_num; ++i)
        {
            if (tid*splited_time_width >= time_width)
                break;
            //i :out id
            short flag = 0;
            unsigned int end_flag = tid*(splited_time_width);//+1);
            short *_dev_out_datas = dev_out_datas + out_start + i * out_width_every;
            unsigned int *_dev_out_times = dev_out_times + out_start + i * out_width_every;
            unsigned int *_dev_valid_width = dev_valid_width + valid_width_start + i * valid_width_size;
            //_dev_out_datas[end_flag] = 1;  //VCD_X
            //_dev_out_times[end_flag] = 0;
            if(tid == 0)
            {
                _dev_out_datas[end_flag] = 1;  //VCD_X
                _dev_out_times[end_flag] = 0;
                ++end_flag;
                pre_out = 1;   //VCD_X;
                last_time = 0;
                _last_time = 0;
            }
            //printf("%d -- out:%d start simulate\n", tid, i);
            for (j = time_start+tid*splited_time_width; j < time_start+(tid+1)*splited_time_width; ++j)
            {
                if (j == time_start+time_width)
                {
                    break;
                }
                if (j == time_start && tid == 0)
                {
                    continue;
                }
                if (j == time_start + tid*splited_time_width && tid != 0 && !flag)
                {
                    j--;
                    flag = 1;
                    //if (bid == 299 && inst_num == 355 && tid < 13 && tid > 10)
                    //    printf("cur_time=%d -- %d\n", dev_times[j], tid);
                }
                //cur_time = dev_times[j];
                cur_time = dev_times[j];               
                
                if (!(j == time_start + tid*splited_time_width - 1 && tid != 0 && flag) && cur_time == _last_time)
                {
                    continue;
                }
                _last_time = cur_time;
                // GetValue at cur_time
                cur_start = in_start+in_num*(j - time_start);
                last_start = in_start+in_num*(j - time_start - 1);
                short *_dev_datas = dev_datas + cur_start;
                short *_l_dev_datas = dev_datas + last_start;
                
                for (k = 0; k < in_num; ++k)
                {
                    _cur_inValues[k] = _dev_datas[k];
                    _last_inValues[k] = _l_dev_datas[k];
                    //if (bid == 299 && tid == 1)
                    //    printf("\tinput %d: %hd -- %d\n", k, _cur_inValues[k], j);
                }
                //if (cur_level == 53 && bid == 0)
                //    printf("time: %u val:%hd, %hd, %hd, %hd(%u) -- %d\n", cur_time, _cur_inValues[0], _cur_inValues[1], _cur_inValues[2], _cur_inValues[3], j, tid); 
                
                // Run Function
                cur_out = 1;    //VCD_X;
                //short *inter_values = new short[(width_every_function - in_num - out_num)*function_num];//(short *)malloc(sizeof(short)*(width_every_function - in_num - out_num)*function_num);
                
                //printf("%d -- (%d)t=%d start run func\n", tid, j, cur_time);
                for (k = 0; k < function_num; ++k)
                {
                    short *_dev_functions = dev_functions + func_start + k * func_width_every;
                    cur_func = _dev_functions[0]; // func_id
                    // input_data
                    func_in_num = 0;
                    for (l = 1; l < width_every_function; ++l)
                    {
                        func_in_num++;
                        if(_dev_functions[l] == -1)
                        {
                            func_in_num = l-1;
                            break;
                        }
                    }
                    func_in_num--;
                    func_flag = _dev_functions[func_in_num+1];
                    //short *input_data = new short[func_in_num];//(short *)malloc(sizeof(short)*func_in_num);
                    pos = _dev_functions[1];
                    //input_data = cur_inValues[inval_start+pos];
                    input_data = _cur_inValues[pos];
                    for (l = 1; l < func_in_num; ++l)
                    {
                        if (cur_func == 9 && l == 2) // udp_mux2
                        {
                            _input_data = input_data1;
                            
                        }
                        pos = _dev_functions[l+1];
                        //input_data1 = cur_inValues[inval_start+pos];
                        input_data1 = _cur_inValues[pos];
                        
                        if (cur_func == 0 || cur_func == 5)  // and
                        {
                            input_data = input_data & input_data1;
                        }
                    
                        else if (cur_func == 1 || cur_func == 4)
                        {
                            input_data = input_data | input_data1;
                        }
                        else if (cur_func == 2 || cur_func == 3)
                        {
                            if (input_data == 1 || input_data1 == 1)
                            {
                                input_data = 1;
                            }
                            else
                                input_data = input_data ^ input_data1;
                        }
                        else if (cur_func == 9 && l == 2)
                        {
                            if(input_data1 != 1)
                                input_data = (input_data1 & _input_data) | ((input_data1  ^ 3) & input_data);
                            else if (_input_data == input_data)
                                input_data = input_data;
                            else
                                input_data = 1;
                        }
                    }
                    if (cur_func == 3 || cur_func == 4 || cur_func == 5 || cur_func == 7)
                    {
                        output_data = input_data ^ 3;   //VCD_1
                    }
                    else
                        output_data = input_data;
                    if (output_data != 0 && output_data != 3 && output_data != 1)
                    {
                        output_data = 1;   //VCD_X
                    }
                    if (func_flag == in_num+i)
                    {
                        cur_out = output_data;
                        break;
                    }
                    _cur_inValues[func_flag] = output_data;
                }
                
                if (j == time_start + tid*splited_time_width-1 && tid != 0 && flag)
                {
                    pre_out = cur_out;
                    //_dev_out_datas[end_flag] = cur_out;
                    last_time = 0;
                    //atomicExch(&dev_out_datas[out_start+i*(global_out_width/out_num)+end_flag], cur_out);
                    continue;
                }
                //printf("%d -- t=%ld start CheckDelayCuda\n", tid, cur_time);
                out_sense = CheckDelayCuda(pre_out, cur_out);
                //printf("%d -- t=%ld success CheckDelayCuda\n", tid, cur_time);
                if (out_sense == 0)
                {
                    continue;
                }
                // add delay
                num_in = 0;
                for (k = 0; k < in_num; ++k)
                {
                    //if (dev_in_datas[in_start+(j - time_start)*in_num+k] != dev_in_datas[in_start+(j -1 - time_start)*in_num+k])
                    //if(cur_inValues[inval_start+k] != last_inValues[lastval_start+k])
                    if(_cur_inValues[k] != _last_inValues[k])
                    {
                        num_in++;
                        changed_id = k;
                    }
                }
                //printf("%d --- t=%ld, delay driving num: %d\n", tid, cur_time, num_in);
                delay_val = 1.0;
                factor = 1.0;
               // printf("%d -- t=%ld start add delay\n", tid, cur_time);
                // only one input driving out
                if (num_in == 1) 
                {
                    delay_valid_id = -1;
                    delay_valid_id1 = -1;
                    delay_valid_id2 = -1;
                    for (k = delay_start; k < delay_start+delay_width; ++k)
                    {
                        if (dev_in_bit[k] == changed_id && dev_delay_edges[k] == 2 && dev_out_bit[k] == i)
                        {
                            delay_valid_id = k;
                            break;
                        }
                        else if (dev_in_bit[k] == changed_id && delay_valid_id1 == -1 && delay_valid_id2 == -1 && dev_out_bit[k] == i)
                        {
                            delay_valid_id1 = k;
                        }
                        else if (dev_in_bit[k] == changed_id && delay_valid_id1 != -1 && delay_valid_id2 == -1 && dev_out_bit[k] == i)
                        {
                            delay_valid_id2 = k;
                        }
                    }
                    if (delay_valid_id == -1)
                    {
                        in_sense = CheckDelayCuda(_last_inValues[changed_id], _cur_inValues[changed_id]);
                        
                        edge = 0;
                        if(in_sense == 1)
                            edge = 0;
                        else
                            edge = 1;
                        if (dev_delay_edges[delay_valid_id1] == edge)
                        {
                            delay_valid_id = delay_valid_id1;
                        }
                        else
                        {
                            delay_valid_id = delay_valid_id2;
                        }
                        
                    }
                    switch(out_sense)
                    {
                        case(1):
                            delay_val = dev_rise_val[delay_valid_id];
                            break;
                        case(2):
                            delay_val = dev_fall_val[delay_valid_id];
                            break;
                        //case(3):
                        //    delay_val = max(dev_rise_val[delay_valid_id], dev_fall_val[delay_valid_id]);
                        //    break;
                        //case(4):
                        //    delay_val = min(dev_rise_val[delay_valid_id], dev_fall_val[delay_valid_id]);
                        //    break;
                        default:
                            delay_val = 1.0;
                            break;
                    }
                }
                // multiple inputs driving out
                else
                {
                    min_val = 100000.0;
                    for (k = 0; k < in_num; ++k)
                    {
                        if(_cur_inValues[k] == _last_inValues[k]) 
                            continue;
                        cur = _cur_inValues[k];
                        last = _last_inValues[k];
                        for (l = delay_start; l < delay_start+delay_width; ++l)
                        {
                            if (dev_in_bit[l] == k && dev_out_bit[l] == i)
                            {
                                if (dev_delay_edges[l] != 2)
                                {
                                    //in_sense = CheckDelayCuda(dev_in_datas[in_start+(j - time_start-1)*in_num+k], dev_in_datas[in_start+(j - time_start)*in_num+k]);
                                    in_sense = CheckDelayCuda(last, cur);
                                    edge = 0;
                                    if(in_sense == 1)
                                        edge = 0;
                                    else
                                        edge = 1;
                                    if (edge != dev_delay_edges[l])
                                    {
                                        continue;
                                    }
                                }
                                switch(out_sense)
                                {
                                    case(1):
                                        min_val = min(dev_rise_val[l], min_val);
                                        break;
                                    case(2):
                                        min_val = min(dev_fall_val[l], min_val);
                                        break;
                                    //case(3):
                                    //    min_val = min(max(dev_rise_val[l], dev_fall_val[l]), min_val);
                                    //    break;
                                    //case(4):
                                    //    min_val = min(min(dev_rise_val[l], dev_fall_val[l]), min_val);
                                    //    break;
                                    default:
                                        min_val = 1.0;
                                        break;
                                }
                            }
                        }
                    }
                    delay_val = (min_val > 99999.0)?1.0:min_val;
                }
                //printf("%d -- t=%ld success add delay\n", tid, cur_time);
                // check time_unit
                //printf("%d -- t=%ld start check time unit\n", tid, cur_time);
                //if(dev_time_unit == 0)
                //    factor = 1e-12;
                //else if(dev_time_unit == 1)
                //    factor = 1e-9;
                //else if(dev_time_unit == 2)
                //    factor = 1e-6;
                //else if(dev_time_unit == 3)
                //    factor = 1e-3;
                //else if(dev_time_unit == 4)
                //    factor = 1.0;
                out_time = cur_time + static_cast<int>(delay_val);//*factor);
                last_value = pre_out;//dev_out_datas[out_start+i*out_width_every+end_flag];
                bool _flag = false;
                //if (cur_level == 0 && bid == 2215)
                //    printf("\ttime: %u val: %hd (last=%u, cur=%u,end_flag=%d) -- %d\n", out_time, cur_out, last_time, cur_time, end_flag, tid);                
                while(last_time >= out_time)
                {
                    //if ((end_flag == tid*(splited_time_width+1)+1 && tid != 0) || end_flag == 0)
                    if ((end_flag == tid*(splited_time_width)+1) || end_flag == 0)
                    {
                        if(end_flag != 0){
                            --end_flag;
                        }
                        _dev_out_datas[end_flag] = -1;
                        _dev_out_times[end_flag] = 0;
                        //end_flag--;
                        _flag = true;
                        break;
                    }
                    --end_flag;
                    _dev_out_datas[end_flag] = -1;
                    _dev_out_times[end_flag] = 0;
                    //atomicExch(&dev_out_datas[out_start+i*out_width_every+end_flag], -1);
                    //atomicExch(&dev_out_times[out_start+i*out_width_every+end_flag], 0);
                    //end_flag--;
                    last_time = _dev_out_times[end_flag - 1];
                    last_value = _dev_out_datas[end_flag - 1];
                    pre_out = last_value;
                }
                //printf("%d -- t=%lld, last=%hd, cur=%hd\n", tid, out_time, last_value, cur_out);
                if (_flag || (last_value != cur_out))
                {
                    //end_flag++;
                    _dev_out_times[end_flag] = out_time;
                    _dev_out_datas[end_flag] = cur_out;
                    last_time = out_time;
                    pre_out = cur_out;
                    ++end_flag;
                }
            }
            _dev_valid_width[tid] = end_flag - tid * splited_time_width;
        }
    }
}

__device__
short CheckDelayCuda(short pre, short cur)
{
    /** delay value --- 1:rise 2:fall 3:max(rise, fall) 4:min(rise, fall) 0:unchanged **/
    if ((pre == 0 && cur != 0) || (cur == 3 && pre != 3))
    {
        return 1;
    }
    else if ((pre == 3 && cur != 3) || (cur == 0 && pre != 0))
    {
        return 2;
    }
    //else if (pre == 2 && cur == 3)
    //{
    //    return 3;
    //}
    //else if (pre == 3 && cur == 2)
    //{
    //    return 4;
    //}
    else
    {
        return 0;
    }   
}

__global__
void DecompressTimesGPU(unsigned int *dev_times, unsigned short *dev_times_compressed, 
    unsigned int *dev_base_start, unsigned int *dev_base_start_start, 
    const unsigned int base_start_total, const unsigned int base_start_start_idx, const unsigned int total_times_sizes)
{
    //const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    unsigned int bid_x = blockIdx.y;
    unsigned int bid_y = blockIdx.x;
    const unsigned int tid = threadIdx.x;
    if(base_start_start_idx > MAX_BLOCKS){
        if(bid_x < base_start_total){
            bid_y = bid_y * 2;
        }
        else{
            bid_y = bid_y * 2 + 1;
            bid_x = bid_x - base_start_total;
        }
        //bid_y = (base_start_start_idx + MAX_BLOCKS - 1) / MAX_BLOCKS * bid_y;
    }
    if(bid_y >= base_start_start_idx){
        return;
    }
    unsigned int base_start_start_val = dev_base_start_start[bid_y];
    unsigned int *_dev_base_start = dev_base_start + base_start_start_val;
    unsigned int tile_num;
    if(bid_y < base_start_start_idx - 1){
        tile_num = dev_base_start_start[bid_y + 1] - base_start_start_val;
    }
    else{
        tile_num = base_start_total - base_start_start_val;
    }
    if(bid_x >= tile_num){
        return;
    }
    unsigned times_start = _dev_base_start[bid_x];
    if(times_start == 0 && base_start_start_val + bid_x > 0){
        return;
    }
    unsigned base = bid_x;
    unsigned int *_dev_times = dev_times + times_start;
    unsigned short *_dev_times_compressed = dev_times_compressed + times_start;
    //unsigned int *_dev_times_compressed = dev_times_compressed + times_start;
    unsigned times_size;
    if(bid_y == base_start_start_idx - 1 && bid_x == tile_num - 1){
        times_size = total_times_sizes - times_start;
    }
    else{
        bool flag = false;
        while(_dev_base_start[bid_x + 1] == 0){
            ++bid_x;
            if(bid_y == base_start_start_idx - 1 && bid_x == tile_num - 1){
                flag = true;
                break;
            }
        }
        if(flag){
            times_size = total_times_sizes - times_start;
        }
        else{
            times_size = _dev_base_start[bid_x + 1] - times_start;
        }
    }
    const unsigned run_width = (times_size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK;
    if(tid * run_width >= times_size){
        return;
    }
    for(unsigned i = tid * run_width; i < times_size && i < (tid + 1) * run_width; ++i){
        _dev_times[i] = static_cast<unsigned int>(_dev_times_compressed[i]) + COMPRESS_BASE * base;
    }
}

__global__
void DecompressGPU(short *dev_in_datas, short *dev_in_data_compressed, 
    unsigned int *dev_step_start_vec, unsigned int *dev_com_step_start_vec, 
    unsigned int *dev_data_val_num, unsigned int blocks_per_signal, unsigned int sigal_num)
{
    const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    const unsigned int _tid = threadIdx.x;
    const unsigned int bid = _bid / blocks_per_signal;
    if(bid >= sigal_num)
    {
        return;
    }
    const unsigned int tid = _tid + (_bid % blocks_per_signal) * N_THREADS_PER_BLOCK;
    const unsigned int data_start = dev_step_start_vec[bid];
    const unsigned int com_data_start = dev_com_step_start_vec[bid];
    const unsigned int data_size = dev_data_val_num[bid];
    const unsigned int com_data_size = (data_size + STATE_NUM - 1) / STATE_NUM;
    const short shift = (3 << 14);
    //const unsigned int tid = _tid + _bid * N_THREADS_PER_BLOCK;
    //if (tid < data_width)
    //{
    //    dev_in_datas[tid] = static_cast<short>(dev_in_data_compressed[tid]);
    //}
    short *_dev_in_data_compressed = dev_in_data_compressed + com_data_start;
    short *_dev_in_datas = dev_in_datas + data_start;
    if (tid < com_data_size)
    {
        short data = _dev_in_data_compressed[tid];
        for(int i = 0; i < STATE_NUM; ++i)
        {
            short cur_data = ((data & shift) >> 14) & 3;
            if(cur_data == UNKNOWN)
            {
                break;
            }
            _dev_in_datas[tid * STATE_NUM + i] = cur_data;
            data = (data << 2);
        }
    }
}

__global__
void CompressGPU(short *dev_in_datas, short *dev_in_data_compressed,
    unsigned int data_width, unsigned int data_width_compress)
{
    const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    const unsigned int _tid = threadIdx.x;
    const unsigned int tid = _tid + _bid * N_THREADS_PER_BLOCK;
    if (tid < data_width_compress)
    {
        const short splited_time_width = COMPRESS_WIDTH;
        int j;//, j_com;
        short temp = 0;
        //bool flag = false;
        //if(tid * splited_time_width >= data_width || tid * (splited_time_width / STATE_NUM) >= data_width_compress){
        //    return;
        //}
        short *_dev_in_datas = dev_in_datas + tid * splited_time_width;
        //short *_dev_in_data_compressed = dev_in_data_compressed + tid * (splited_time_width / STATE_NUM);
        for(j = 0; j < splited_time_width; ++j){
            short data = _dev_in_datas[j];
            if(data == -1 || j + tid * splited_time_width >= data_width){
                temp = (temp << 2);
                temp += UNKNOWN;   //unknown
            }
            else{
                temp = (temp << 2);
                temp += data;
            }
        }
        dev_in_data_compressed[tid] = temp;
        /*for (j = 0, j_com = 0; 
            j_com < (splited_time_width / STATE_NUM) && j < splited_time_width; 
            ++j_com)
        {
            if(j_com + tid * (splited_time_width / STATE_NUM) == data_width_compress){
                break;
            }
            temp = 0;
            for(int k = 0; k < STATE_NUM; ++k, ++j){
                data = _dev_in_datas[j];
                if(j + tid * splited_time_width >= data_width || data == -1){
                    temp = (temp << 2);
                    temp += UNKNOWN;   //unknown
                }
                else{
                    temp = (temp << 2);
                    temp += data;
                }
            }
            _dev_in_data_compressed[j_com] = temp;
        }*/
    }
}

__global__
void ProcessConflictCuda(int cur_level,int blocks_per_inst, short *dev_out_datas, unsigned int *dev_out_times,
                int dev_time_unit, short *dev_data_out_num,
                short *dev_out_splited_width, unsigned int *dev_output_start, unsigned int *dev_output_size, 
                unsigned int *dev_data_in_num_start,
                unsigned int *dev_valid_width, unsigned int *dev_valid_width_start, unsigned int *dev_valid_width_size, 
                short *dev_valid_width_flag)
{
    const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    const unsigned int _tid = threadIdx.x;
    if (_tid < N_THREADS_PER_BLOCK)
    {
        const unsigned int bid = _bid / blocks_per_inst;
        const unsigned int tid = _tid + (_bid % blocks_per_inst) * N_THREADS_PER_BLOCK;
        //__shared__ int dev_in_times[12288];
        //printf("tid:%d, time_width:%d, time_start:%d\n", tid, time_width, time_start);
        const unsigned int _start = dev_data_in_num_start[cur_level];
        //out data flag
        const short out_num = dev_data_out_num[_start+bid];
        //begin point
        const unsigned int out_start = dev_output_start[bid];//bid*global_out_width;
        const unsigned int out_width_every = dev_output_size[bid];//global_out_width/out_num;
        const unsigned int valid_width_start = dev_valid_width_start[bid];//bid*global_out_width;
        const unsigned int valid_width_size = dev_valid_width_size[bid];//global_out_width/out_num;
        //printf("%d start calculate\n", tid);
        unsigned int end_flag, flag;
        unsigned int cur_time, last_time;
        unsigned int i,j,k,l;
        short cur_data, last_data;
        //__shared__ int _time_width[N_THREADS_PER_BLOCK];

        short splited_time_width = dev_out_splited_width[bid];
        if (tid == 0 || tid*splited_time_width >= out_width_every)
            return;
        for (i = 0; i < out_num; ++i)
        {
            //i :out id
            short *_dev_out_datas = dev_out_datas + out_start + i * out_width_every;
            unsigned int *_dev_out_times = dev_out_times + out_start + i * out_width_every;
            unsigned int *_dev_valid_width = dev_valid_width + valid_width_start + i * valid_width_size;
            short *_dev_valid_width_flag = dev_valid_width_flag + valid_width_start + i * valid_width_size;
            //set 0
            //_dev_out_datas[tid * splited_time_width] = -1;
            //_dev_out_times[tid * splited_time_width] = 0;
            last_time = 0;
            last_data = 1;
            for(j = (tid - 1) * splited_time_width; j < tid * splited_time_width && _dev_out_datas[j] != -1; ++j)
            {
                last_time = _dev_out_times[j];
                last_data = _dev_out_datas[j];
            }
            end_flag = j - 1;
            if (end_flag == (tid - 1) * splited_time_width - 1)
            {
                continue;
            }
            
            
            //
            flag = tid * splited_time_width;// + 1;
            cur_time = _dev_out_times[flag];
            cur_data = _dev_out_datas[flag];
            
            if (cur_data == -1 || cur_time == 0)
            {
                continue;
            }
            //last_time = _dev_out_times[end_flag];
            //if (bid < 10)
            //{
            //    printf("end_flag:%d, last_time:%u, cur_time:%u -- %d, %d\n", end_flag, last_time, cur_time, tid, bid);
            //}
            //if (cur_level ==30 && bid == 0 && cur_time > 2876754000 && cur_time < 2876754600 && i == 1)
            //    printf("end=%u, last=%u(%hd), cur=%u(%hd) -- %u\n",end_flag,last_time,_dev_out_datas[end_flag], cur_time,cur_data, tid);
            bool _flag = false;
            while(last_time >= cur_time)
            {
                if (end_flag == (tid-1)*splited_time_width)//+1) && tid > 1)
                {
                    _dev_out_datas[end_flag] = -1;
                    _dev_out_times[end_flag] = 0;
                    end_flag--;
                    --_dev_valid_width[tid - 1];
                    _flag = true;
                    break;
                }
                //if (cur_level ==28 && bid == 0 && tid == 69632 && i == 1)
                //    printf("end=%u, last=%u(%hd), cur=%u(%hd) -- %d\n",end_flag,last_time,_dev_out_datas[end_flag], cur_time,cur_data, tid);
                _dev_out_datas[end_flag] = -1;
                _dev_out_times[end_flag] = 0;
                end_flag--;
                --_dev_valid_width[tid - 1];
                last_time = _dev_out_times[end_flag];        
                last_data = _dev_out_datas[end_flag];        
            }
            if (cur_data == last_data && _flag == false)
            {
                _dev_out_datas[flag] = -1;
                _dev_out_times[flag] = 0;
                _dev_valid_width_flag[tid] = 1;
            }
        }
    }
}

__global__
void UpdateValidWidthGPU(int cur_level,int blocks_per_inst, short *dev_out_datas,
                short *dev_data_out_num,
                short *dev_out_splited_width, unsigned int *dev_output_start, unsigned int *dev_output_size, 
                unsigned int *dev_data_in_num_start,
                unsigned int *dev_valid_width, unsigned int *dev_valid_width_start, unsigned int *dev_valid_width_size, 
                short *dev_valid_width_flag)
{
    const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    const unsigned int _tid = threadIdx.x;
    if (_tid < N_THREADS_PER_BLOCK)
    {
        const unsigned int bid = _bid / blocks_per_inst;
        const unsigned int tid = _tid + (_bid % blocks_per_inst) * N_THREADS_PER_BLOCK;
        //__shared__ int dev_in_times[12288];
        //printf("tid:%d, time_width:%d, time_start:%d\n", tid, time_width, time_start);
        const unsigned int _start = dev_data_in_num_start[cur_level];
        //out data flag
        const short out_num = dev_data_out_num[_start+bid];
        //begin point
        const unsigned int out_start = dev_output_start[bid];//bid*global_out_width;
        const unsigned int out_width_every = dev_output_size[bid];//global_out_width/out_num;
        const unsigned int valid_width_start = dev_valid_width_start[bid];//bid*global_out_width;
        const unsigned int valid_width_size = dev_valid_width_size[bid];//global_out_width/out_num;
        //printf("%d start calculate\n", tid);
        unsigned int i;
        //__shared__ int _time_width[N_THREADS_PER_BLOCK];

        short splited_time_width = dev_out_splited_width[bid];
        if (tid == 0 || tid*splited_time_width >= out_width_every)
            return;
        for (i = 0; i < out_num; ++i)
        {
            //i :out id
            short *_dev_out_datas = dev_out_datas + out_start + i * out_width_every;
            unsigned int *_dev_valid_width = dev_valid_width + valid_width_start + i * valid_width_size;
            short *_dev_valid_width_flag = dev_valid_width_flag + valid_width_start + i * valid_width_size;

            unsigned int flag = tid * splited_time_width;// + 1;
            if(_dev_valid_width_flag[tid] == 1){
                --_dev_valid_width[tid];
            }
            
        }
    }
}

__global__
void UpdateOutInfoGPU(unsigned int *dev_prefix_sum_of_valid_width, int cur_level,int inst_num,
                short *dev_data_out_num,
                short *dev_out_splited_width, unsigned int *dev_output_start, unsigned int *dev_output_size, 
                unsigned int *dev_data_in_num_start,
                unsigned int *dev_valid_width, unsigned int *dev_valid_width_start, unsigned int *dev_valid_width_size,
                unsigned int *dev_out_width)
{
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < inst_num)
    {
        const unsigned int _start = dev_data_in_num_start[cur_level];
        //out data flag
        const short out_num = dev_data_out_num[_start+tid];
        //begin point
        const unsigned int valid_width_start = dev_valid_width_start[tid];//bid*global_out_width;
        
        const unsigned int valid_width_size = dev_valid_width_size[tid];//global_out_width/out_num;
        //printf("%d start calculate\n", tid);
        //__shared__ int _time_width[N_THREADS_PER_BLOCK];

        unsigned int *_dev_prefix_sum_of_valid_width = dev_prefix_sum_of_valid_width + valid_width_start;
        unsigned int start = _dev_prefix_sum_of_valid_width[0];
        dev_output_start[tid] = start;
        for (int i = 0; i < out_num; ++i)
        {
            if(tid == inst_num - 1 && i == out_num - 1){
                unsigned int *_dev_valid_width = dev_valid_width + valid_width_start;
                *dev_out_width = _dev_prefix_sum_of_valid_width[valid_width_size*(i+1) - 1] + _dev_valid_width[valid_width_size*(i+1) - 1];
                dev_output_size[tid * MAX_OUT_NUM + i] = *dev_out_width - start;
            }
            else{
                dev_output_size[tid * MAX_OUT_NUM + i] = _dev_prefix_sum_of_valid_width[valid_width_size*(i+1)] - start;
                start = _dev_prefix_sum_of_valid_width[valid_width_size];
            }
        }
        
    }
}

__global__
void RemoveInvalidStatesGPU(unsigned int *dev_out_times_new, short *dev_out_datas_new,
                int cur_level, int blocks_per_inst, short *dev_out_datas, unsigned int *dev_out_times,
                short *dev_data_out_num,
                short *dev_out_splited_width, unsigned int *dev_output_start, unsigned int *dev_output_size, 
                unsigned int *dev_data_in_num_start,
                unsigned int *dev_valid_width, unsigned int *dev_valid_width_start, unsigned int *dev_valid_width_size,
                unsigned int *dev_prefix_sum_of_valid_width)
{

    const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    const unsigned int _tid = threadIdx.x;
    if (_tid < N_THREADS_PER_BLOCK)
    {
        const unsigned int bid = _bid / blocks_per_inst;
        const unsigned int tid = _tid + (_bid % blocks_per_inst) * N_THREADS_PER_BLOCK;
        //__shared__ int dev_in_times[12288];
        //printf("tid:%d, time_width:%d, time_start:%d\n", tid, time_width, time_start);
        const unsigned int _start = dev_data_in_num_start[cur_level];
        //out data flag
        const short out_num = dev_data_out_num[_start+bid];
        //begin point
        const unsigned int out_start = dev_output_start[bid];//bid*global_out_width;
        const unsigned int out_width_every = dev_output_size[bid];//global_out_width/out_num;

        const unsigned int valid_width_start = dev_valid_width_start[bid];//bid*global_out_width;
        const unsigned int valid_width_size = dev_valid_width_size[bid];//global_out_width/out_num;
        //printf("%d start calculate\n", tid);
        //__shared__ int _time_width[N_THREADS_PER_BLOCK];

        short splited_time_width = dev_out_splited_width[bid];
        if (tid*splited_time_width >= out_width_every)
            return;
        for (int i = 0; i < out_num; ++i)
        {
            //i :out id
            short *_dev_out_datas = dev_out_datas + out_start + i * out_width_every;
            unsigned int *_dev_out_times = dev_out_times + out_start + i * out_width_every;
            

            unsigned int *_dev_valid_width = dev_valid_width + valid_width_start + i * valid_width_size;
            unsigned int *_dev_prefix_sum_of_valid_width = dev_prefix_sum_of_valid_width + valid_width_start + i * valid_width_size;
            unsigned int _valid_width = _dev_valid_width[tid];
            unsigned int _valid_start = _dev_prefix_sum_of_valid_width[tid];
            short *_dev_out_datas_new = dev_out_datas_new + _valid_start;
            unsigned int *_dev_out_times_new = dev_out_times_new + _valid_start;

            for (unsigned j = tid*splited_time_width, k = 0; k < _valid_width && j < (tid+1)*splited_time_width; ++j){
                if(j == out_width_every){
                    break;
                }
                if(_dev_out_datas[j] == -1){
                    continue;
                }
                _dev_out_times_new[k] = _dev_out_times[j];
                _dev_out_datas_new[k] = _dev_out_datas[j];
                ++k;
            }            
        }
    }

}

void checkError(hipError_t error, std::string msg) {
    if (error != hipSuccess) {
        printf("%s: %d\n", msg.c_str(), error);
        cout << hipGetErrorString(error) << endl;
        exit(-1);
    }
}

__device__ int cu_abs(int i, int j)
{
    if (i <= j){
        return (j - i) >> 1;
    }
    else{
        return (i - j) >> 1;
    }
}
__global__ void PreCudaMerge(unsigned int j, unsigned int *dev_times, short *dev_datas, 
    short *dev_in_datas, int _blocks_per_inst, int cur_level, int inst_id_start, int inst_num_start,int inst_num_end, unsigned int *dev_total_times_start,
    unsigned int *dev_input_start, unsigned int *dev_data_val_num, unsigned int *dev_val_num_start, unsigned int *dev_data_in_num_start, short *dev_data_in_num,
    unsigned int *dev_temp_times, short *dev_temp_datas) {
    const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    const unsigned int _tid = threadIdx.x;
    const unsigned int bid = _bid / _blocks_per_inst + inst_id_start;
    if (bid >= inst_num_end || bid < inst_num_start)
    {
        return;
    }
    
    const unsigned int tid = _tid + (_bid % _blocks_per_inst) * _N_THREADS_PER_BLOCK;
    
    const unsigned int _start = dev_data_in_num_start[cur_level];
    const short in_num = dev_data_in_num[_start+bid];
    const unsigned int val_num_start = dev_val_num_start[bid];
    const unsigned int A_start = dev_total_times_start[bid];
    const unsigned int A_data_start = dev_input_start[bid];
    unsigned length_A = dev_data_val_num[val_num_start];
    //if(cur_level == 0 && bid == 2215 && tid == 0){
    //    printf("t=%u\n", *(dev_times + A_start + length_A / 2));
    //}
    for (int i = 1; i < in_num; ++i)
    {
        if (i < j)
        {
            length_A += dev_data_val_num[val_num_start+i];
        }
        //total_length += dev_data_val_num[val_num_start+i];
    }
    const unsigned int B_start = A_start + length_A;
    short *A_datas = dev_in_datas + A_start;// - dev_total_times_start[inst_num_start];
    short *D_datas = dev_datas + A_data_start;
    if (in_num == 1 && j == 1 && tid < length_A)
    {
        D_datas[tid] = A_datas[tid];
    }
    unsigned length_B = dev_data_val_num[val_num_start+j];
    if (j < in_num && tid < length_A + length_B)
    {
        unsigned int *A = dev_times + A_start;
        unsigned int *B = dev_times + B_start;
        
        short *B_datas = dev_in_datas + B_start;// - dev_total_times_start[inst_num_start];
        short *temp_datas = dev_temp_datas + A_data_start - dev_input_start[inst_num_start];
        unsigned int *temp_times = dev_temp_times + A_start - dev_total_times_start[inst_num_start];
        //if(cur_level == 1 && _bid == 0 && _tid < 3)
        //    printf("initial done!\n");
        duo K, P, Q;
        if (tid > length_A) {
            K.x = tid - length_A;
            K.y = length_A;
            P.x = length_A;
            P.y = tid - length_A;
        }else {
            K.x = 0;
            K.y = tid;
            P.x = tid;
            P.y = 0;
        }
        while (true) {
            int offset = cu_abs(K.y, P.y);
            Q.x = K.x + offset;
            Q.y = K.y - offset;
            if (Q.y >= 0 && Q.x <= length_B && (Q.y == length_A || Q.x == 0 || A[Q.y] > B[Q.x - 1])){
                if (Q.x == length_B || Q.y == 0 || A[Q.y - 1] <= B[Q.x]) {
                    if (Q.y < length_A && (Q.x == length_B || A[Q.y] <= B[Q.x])){
                        temp_times[tid] = A[Q.y];
                        //store data
                        if (j == 1)
                        {
                            D_datas[tid*2] = A_datas[Q.y];
                        }
                        else
                        {
                            for (int i = 0; i < j; ++i)
                            {
                                //dev_datas[A_data_start+tid*(j+1)+i] = dev_temp_datas[A_data_start+Q.y*j+i];
                                D_datas[tid*(j+1)+i] = temp_datas[Q.y*j+i];
                            }
                        }
                        if (A[Q.y] == B[Q.x]){
                            //dev_datas[A_data_start+tid*(j+1)+j] = dev_in_datas[B_start+Q.x];
                            D_datas[tid*(j+1)+j] = B_datas[Q.x];
                        }
                        else if(Q.x == 0){
                            D_datas[tid*(j+1)+j] = 1;   //VCD_X
                        }
                        else{
                            D_datas[tid*(j+1)+j] = B_datas[Q.x - 1];
                        }
                    }
                    else{
                        temp_times[tid] = B[Q.x];
                        //store data
                        if (j == 1)
                        {
                            if(Q.y == 0){
                                D_datas[tid*2] = 1;
                            }
                            else{
                                D_datas[tid*2] = A_datas[Q.y - 1];
                            }
                        }
                        else
                        {
                            if(Q.y == 0){
                                for (int i = 0; i < j; ++i)
                                {
                                    D_datas[tid*(j+1)+i] = 1;
                                }
                            }
                            else{
                                for (int i = 0; i < j; ++i)
                                {
                                    D_datas[tid*(j+1)+i] = temp_datas[(Q.y-1)*j+i];
                                }
                            }
                        }
                        D_datas[tid*(j+1)+j] = B_datas[Q.x];
                    }
                    break;
                }
                else {
                    K.x = Q.x + 1;
                    K.y = Q.y - 1;
                }
            }
            else {
                P.x = Q.x - 1;
                P.y = Q.y + 1;
            }
        }
    }
}

__global__ void CudaMerge(unsigned int j, unsigned int *dev_times, short *dev_datas, 
    int _blocks_per_inst, int cur_level, int inst_id_start, int inst_num_start,int inst_num_end, unsigned int *dev_total_times_start,
    unsigned int *dev_input_start, unsigned int *dev_data_val_num, unsigned int *dev_val_num_start, unsigned int *dev_data_in_num_start, short *dev_data_in_num,
    unsigned int *dev_temp_times, short *dev_temp_datas) {
    const unsigned int _bid = blockIdx.x + blockIdx.y * gridDim.x;
    const unsigned int _tid = threadIdx.x;
    const unsigned int bid = _bid / _blocks_per_inst + inst_id_start;
    if (bid >= inst_num_end || bid < inst_num_start)
    {
        return;
    }
    const unsigned int tid = _tid + (_bid % _blocks_per_inst) * _N_THREADS_PER_BLOCK;
    const unsigned int _start = dev_data_in_num_start[cur_level];
    const short in_num = dev_data_in_num[_start+bid];
    const unsigned int val_num_start = dev_val_num_start[bid];
    const unsigned int A_start = dev_total_times_start[bid];
    const unsigned int A_data_start = dev_input_start[bid];
    unsigned length_A = dev_data_val_num[val_num_start];
    //unsigned total_length = dev_data_val_num[val_num_start];
    for (int i = 1; i < in_num; ++i)
    {
        if (i < j)
        {
            length_A += dev_data_val_num[val_num_start+i];
        }
        //total_length += dev_data_val_num[val_num_start+i];
    }
    unsigned length_B = dev_data_val_num[val_num_start+j];
    if (j < in_num && tid < length_A + length_B)
    {
        unsigned int *times = dev_times + A_start;
        unsigned int *temp_times = dev_temp_times + A_start - dev_total_times_start[inst_num_start];
        short *temp_datas = dev_temp_datas + A_data_start - dev_input_start[inst_num_start];
        short *datas = dev_datas + A_data_start;
        times[tid] = temp_times[tid];
        if (j < in_num - 1){
            for (int i = 0; i <= j; ++i)
            {
                temp_datas[tid*(j+1)+i] = datas[tid*(j+1)+i];
            }
        }
    }
}

/*void processCore(int left, int right, int inst_num, inter &_inter, VCDParser &parser, VCDTime sim_start, VCDTime sim_end,
    unsigned int *total_input_times, char *input_datas,
    std::vector<int> &cur_level, std::vector<unsigned int*> &out_starts, std::vector<unsigned int*> &out_sizes,
    std::vector<unsigned int*> &out_times, std::vector<short*> &out_values, std::map<std::string, TimedValues*> &pinbitValues,
    std::vector<unsigned int> &total_times_start, std::unordered_map<std::string, bool> &cur_flag,
    std::unordered_map<std::string, std::vector<unsigned long long>> &temp_Result)*/
void processCore(int left, int right, int size, VCDTime sim_start, VCDTime sim_end,
    unsigned int *total_input_times, short *input_datas,
    std::vector<std::string> &name_vec, std::vector<unsigned int*> &time_vec, std::vector<short*> &value_vec, 
    unsigned int *step_vec, std::vector<bool> &is_init_vec, std::vector<short> &offset_vec, 
    std::vector<unsigned int> &step_start_vec, std::vector<unsigned int> &com_step_start_vec, 
    std::vector<bool> &cur_flag,
    std::unordered_map<std::string, std::vector<unsigned long long>> &temp_Result)
{
    //debug
    //bool flag = false;
    //
    for (int j = left; j < right && j < size; ++j)
    {
        string _name = name_vec[j];
        //if(_name == "tile_ICCADs_core_ICCADs_div_ICCADs_intadd_10_ICCADs_SUM_6_"){
        //    flag = true;
        //}
        unsigned int *pos = time_vec[j];
        short *pos_val = value_vec[j];
        unsigned int step = step_vec[j];
        bool is_init = is_init_vec[j];
        short offset = offset_vec[j];
        unsigned int sum = 0;
        unsigned int sum_compress = 0;
        unsigned int data_width = step_start_vec[j];//total_times_start[j];
        unsigned int data_width_compress = com_step_start_vec[j];
        bool _cur_flag = cur_flag[j];
        if (is_init)
        {
            unsigned long long time_1=0;
            unsigned long long time_0=0;
            unsigned long long time_x=0;
            unsigned long long time_z=0;
            unsigned long long last_time = sim_start;
            VCDBit last_state = VCD_X;
            short data = 0;
            unsigned k;
            for (k = 0; k < step; ++k)
            {
                total_input_times[sum+data_width] = pos[k];
                if(k > 0 && k % 8 == 0){
                    input_datas[sum_compress+data_width_compress] = data;
                    ++sum_compress;
                    data = 0;
                }
                data = (data << 2);
                data += pos_val[k];
                
                //input_datas[sum+data_width] = static_cast<char>(pos_val[k]);
                ++sum;
                if(_cur_flag){
                    continue;
                }
                VCDTime cur_t = pos[k];
                VCDBit cur_val = VCDBit(pos_val[k]);
                if (cur_t <= sim_start)
                {
                    last_state = cur_val;
                    continue;
                }
                if (cur_t > sim_end)
                {
                    continue;
                }
                if (last_state == VCD_X)
                {
                    time_x += cur_t - last_time;
                }
                else if (last_state == VCD_0)
                {
                    time_0 += cur_t - last_time;
                }
                else if (last_state == VCD_1)
                {
                    time_1 += cur_t - last_time;
                }
                else if (last_state == VCD_Z)
                {
                    time_z += cur_t - last_time;
                }
                last_time = cur_t;
                last_state = cur_val;
            }
            while(k % 8){
                data = (data << 2);
                data += UNKNOWN;
                ++k;
            }
            input_datas[sum_compress+data_width_compress] = data;
            //++sum_compress;
            
            if (!_cur_flag && last_time < sim_end)
            {
                if (last_state == VCD_X)
                {
                    time_x += sim_end - last_time;
                }
                else if (last_state == VCD_0)
                {
                    time_0 += sim_end - last_time;
                }
                else if (last_state == VCD_1)
                {
                    time_1 += sim_end - last_time;
                }
                else if (last_state == VCD_Z)
                {
                    time_z += sim_end - last_time;
                }
            }
            if(!_cur_flag){
                std::vector<unsigned long long> res = {time_0, time_1, time_x+time_z};
                Result[_name] = res;
                temp_Result[_name] = res;
            }
        }
        else
        {
            unsigned long long time_1=0;
            unsigned long long time_0=0;
            unsigned long long time_x=0;
            unsigned long long time_z=0;
            unsigned long long last_time = sim_start;
            VCDBit last_state = VCD_X;
            // find real start
            unsigned pos_val_start = 0;
            short start_data = pos_val[pos_val_start];
            short data;
            short shift = (3 << 14);
            int o;
            for(o = 0; o < offset; ++o){
                start_data = (start_data << 2);
            }
            short temp_data = 0;
            unsigned k;
            for (k = 0; k < step; ++k)
            {
                short data = ((start_data & shift) >> 14) & 3;
                ++o;
                start_data = (start_data << 2);
                if(o == STATE_NUM){
                    ++pos_val_start;
                    o = 0;
                    start_data = pos_val[pos_val_start];
                }
                total_input_times[sum+data_width] = pos[k];
                if(k > 0 && k % 8 == 0){
                    input_datas[sum_compress+data_width_compress] = temp_data;
                    ++sum_compress;
                    temp_data = 0;
                }
                temp_data = (temp_data << 2);
                temp_data += data;

                //input_datas[sum+data_width] = static_cast<char>(data);
                ++sum;
                if(_cur_flag){
                    continue;
                }
                VCDTime cur_t = pos[k];
                VCDBit cur_val = VCDBit(data);//pos_val[k]);
                if (cur_t <= sim_start)
                {
                    last_state = cur_val;
                    continue;
                }
                if (cur_t > sim_end)
                {
                    continue;
                }
                if (last_state == VCD_X)
                {
                    time_x += cur_t - last_time;
                }
                else if (last_state == VCD_0)
                {
                    time_0 += cur_t - last_time;
                }
                else if (last_state == VCD_1)
                {
                    time_1 += cur_t - last_time;
                }
                else if (last_state == VCD_Z)
                {
                    time_z += cur_t - last_time;
                }
                last_time = cur_t;
                last_state = cur_val;
            }
            while(k % 8){
                temp_data = (temp_data << 2);
                temp_data += UNKNOWN;
                ++k;
            }
            input_datas[sum_compress+data_width_compress] = temp_data;
            if (!_cur_flag && last_time < sim_end)
            {
                if (last_state == VCD_X)
                {
                    time_x += sim_end - last_time;
                }
                else if (last_state == VCD_0)
                {
                    time_0 += sim_end - last_time;
                }
                else if (last_state == VCD_1)
                {
                    time_1 += sim_end - last_time;
                }
                else if (last_state == VCD_Z)
                {
                    time_z += sim_end - last_time;
                }
            }
            if(!_cur_flag){
                std::vector<unsigned long long> res = {time_0, time_1, time_x+time_z};
                Result[_name] = res;
                temp_Result[_name] = res;
            }
        }
    }
}

int main(int argc, char const *argv[])
{
    std::string vcdFilePath, database_path, saif_out, dumpon_time_str, dumpoff_time_str;
    int dumpon_time, dumpoff_time;
    if (argc == 6)
    {
        vcdFilePath = argv[2];
        database_path = argv[1];
        dumpon_time_str = argv[3];
        dumpoff_time_str = argv[4];
        saif_out = argv[5];
    }
    else
    {
        std::cout << "[USAGE] ./simulate [intermediate_file] vcd_path dumpon_time(ps) dumpoff_time(ps) saif_out" << std::endl;
        exit(-1);
    }
    //PreProcess processor;
    ofstream outf(saif_out);
    if (!outf)
    {
        cout << "File " << saif_out << " Open Error!" << endl;
        cout << "Exit!" << endl;
        exit(-1);
    }
    VCDTime sim_start = stoull(dumpon_time_str);
    VCDTime sim_end = stoull(dumpoff_time_str);

    std::vector<unsigned int> data_in_num_start;
    std::vector<unsigned int> data_out_num_start;
    std::vector<unsigned int> delay_start;
    std::vector<unsigned int> delay_width;
    std::vector<unsigned int> functions_start;
    std::vector<unsigned int> functions_width;

    std::vector<short> data_in_num;
    std::vector<short> data_out_num;
    //std::vector<unsigned int> val_num_start;
    std::vector<short> delay_val_num;
    std::vector<short> functions_func_num;
    std::vector<short> functions_val_num;
    std::vector<short> host_delay_edges;
    std::vector<short> host_in_bit;
    std::vector<short> host_out_bit;
    std::vector<float> host_rise_val;
    std::vector<float> host_fall_val;

    std::vector<short> host_functions;
    auto start_total = std::chrono::steady_clock::now();
    inter _inter;
    ifstream ifs(database_path.c_str());
    boost::archive::text_iarchive ia(ifs);
    ia & _inter;
    cout << "new:" << endl;
    //debug
    cout << _inter.levels.size() << endl;
    cout << _inter.p.instances.size() << endl;
    //
    ifs.close();
    hipDeviceProp_t cudade;
    /*hipGetDeviceProperties(&cudade,0);
    std::cout << "GPU型号： " << cudade.name << std::endl;
    std::cout << "每块全局内存存储容量（B）: " << cudade.totalGlobalMem << std::endl;
    std::cout << "每块共享内存存储容量（B）: "<< cudade.sharedMemPerBlock << std::endl;
    std::cout << "每块寄存器数量: " << cudade.regsPerBlock << std::endl;
    std::cout << "WarpSize：  " << cudade.warpSize << std::endl;
    std::cout << "最大内存复制步长：  " << cudade.memPitch << std::endl;
    std::cout << "每块最大线程数量：  " << cudade.maxThreadsPerBlock << std::endl;
    std::cout << "线程块三维： " << cudade.maxThreadsDim << std::endl;
    std::cout << "线程格三维: " << cudade.maxGridSize << std::endl;
    //std::cout << "计算核心时钟频率（kHz）：  " << cudade.clockRate << std::endl;
    //std::cout << "常量存储容量：  " << cudade.totalConstMem << std::endl;
    //std::cout << "次计算能力（小数点后的值）： " << cudade.minor << std::endl;
    //std::cout << "纹理对齐要求: " << cudade.textureAlignment << std::endl;
    //std::cout << "绑定到等步长内存的纹理满足的要求: " << cudade.texturePitchAlignment << std::endl;
    //std::cout << "GPU是否支持并发内存复制和kernel执行: " << cudade.deviceOverlap << std::endl;
    std::cout << "SMX数量：  " << cudade.multiProcessorCount << std::endl;
    //std::cout << "是否有运行时限制：  " << cudade.kernelExecTimeoutEnabled << std::endl;
    //std::cout << "设备是否集成（否则独立）：  " << cudade.integrated << std::endl;
    //std::cout << "可否对主机内存进行映射： " << cudade.canMapHostMemory << std::endl;
    std::cout << "计算模式: " << cudade.computeMode << std::endl;
    //std::cout << "最大1D纹理尺寸：  " << cudade.maxTexture1D << std::endl;
    //std::cout << "线性内存相关的最大1D纹理尺寸：  " << cudade.maxTexture1DLinear << std::endl;
    //std::cout << "最大2D纹理维度：  " << cudade.maxTexture2D << std::endl;
    //std::cout << "最大2D纹理维度（width,height,pitch）： " << cudade.maxTexture2DLinear << std::endl;
    //std::cout << "纹理聚集时的最大纹理维度: " << cudade.maxTexture2DGather << std::endl;
    /*std::cout << "最大3D纹理维度: " << cudade.maxTexture3D << std::endl;
    std::cout << "最大立方图纹理维度: " << cudade.maxTextureCubemap << std::endl;
    std::cout << "最大1D分层纹理维度：  " << cudade.maxTexture1DLayered << std::endl;
    std::cout << "最大2D分层纹理维度：  " << cudade.maxTexture2DLayered << std::endl;
    std::cout << "最大立方图分层纹理维度：  " << cudade.maxTextureCubemapLayered << std::endl;
    std::cout << "最大1D表面尺寸： " << cudade.maxSurface1D << std::endl;
    std::cout << "主计算能力（小数点前的值）：  " << cudade.major << std::endl;
    std::cout << "最大2D表面维度: " << cudade.maxSurface2D << std::endl;
    std::cout << "最大3D表面维度：  " << cudade.maxSurface3D << std::endl;
    std::cout << "最大1D分层表面维度：  " << cudade.maxSurface1DLayered << std::endl;
    std::cout << "最大2D分层表面维度：  " << cudade.maxSurface2DLayered << std::endl;
    std::cout << "最大立方图表面维度： " << cudade.maxSurfaceCubemap << std::endl;
    std::cout << "表面对齐要求: " << cudade.surfaceAlignment << std::endl;
    std::cout << "设备能并发的kernel数量: " << cudade.concurrentKernels << std::endl;
    std::cout << "是否打开ECC校验: " << cudade.ECCEnabled << std::endl;
    std::cout << "PCI总线ID：  " << cudade.pciBusID << std::endl;
    std::cout << "PCI设备ID：  " << cudade.pciDeviceID << std::endl;
    std::cout << "PCI域ID：  " << cudade.pciDomainID << std::endl;
    std::cout << "是否支持TCC（Tesla集群）： " << cudade.tccDriver << std::endl;
    std::cout << "异步引擎数量: " << cudade.asyncEngineCount << std::endl;
    std::cout << "主机和设备共享同一地址空间：  " << cudade.unifiedAddressing << std::endl;
    std::cout << "存储时钟频率：  " << cudade.memoryClockRate << std::endl;
    std::cout << "Global memory 总线带宽：  " << cudade.memoryBusWidth << std::endl;
    std::cout << "L2 Cache尺寸（B)：  " << cudade.l2CacheSize << std::endl;*/
    //std::cout << "每个SMX驻留的最大线程数量：  " << cudade.maxThreadsDim << std::endl;
    hipSetDevice(0);
    hipError_t err;
    // malloc global parameters
    auto start_malloc = std::chrono::steady_clock::now();
    unsigned int *dev_data_in_num_start;
    unsigned int *dev_data_out_num_start;
    unsigned int *dev_delay_start;
    unsigned int *dev_delay_width;
    unsigned int *dev_functions_start;
    unsigned int *dev_functions_width;
    //unsigned int *dev_val_num_start;
    short *dev_data_in_num;
    short *dev_data_out_num;
    short *dev_delay_val_num;
    short *dev_functions_func_num;
    short *dev_functions_val_num;
    short *dev_delay_edges;
    short *dev_in_bit;
    short *dev_out_bit;
    float *dev_rise_val;
    float *dev_fall_val;
    short *dev_functions;
    err=hipMalloc((void **)&dev_data_in_num_start, sizeof(unsigned int)*(_inter.data_in_num_start).size());
    err=hipMalloc((void **)&dev_data_out_num_start, sizeof(unsigned int)*(_inter.data_out_num_start).size());
    err=hipMalloc((void **)&dev_delay_start, sizeof(unsigned int)*(_inter.delay_start).size());
    err=hipMalloc((void **)&dev_delay_width, sizeof(unsigned int)*(_inter.delay_width).size());
    err=hipMalloc((void **)&dev_functions_start, sizeof(unsigned int)*(_inter.functions_start).size());
    err=hipMalloc((void **)&dev_functions_width, sizeof(unsigned int)*(_inter.functions_width).size());
    //err=hipMalloc((void **)&dev_val_num_start, sizeof(unsigned int)*val_num_start.size());
    err=hipMalloc((void **)&dev_data_in_num, sizeof(short)*(_inter.data_in_num).size());
    err=hipMalloc((void **)&dev_data_out_num, sizeof(short)*(_inter.data_out_num).size());
    err=hipMalloc((void **)&dev_delay_val_num, sizeof(short)*(_inter.delay_val_num).size());
    err=hipMalloc((void **)&dev_functions_func_num, sizeof(short)*(_inter.functions_func_num).size());
    err=hipMalloc((void **)&dev_functions_val_num, sizeof(short)*(_inter.functions_val_num).size());
    err=hipMalloc((void **)&dev_delay_edges, sizeof(short)*(_inter.host_delay_edges).size());
    err=hipMalloc((void **)&dev_in_bit, sizeof(short)*(_inter.host_in_bit).size());
    err=hipMalloc((void **)&dev_out_bit, sizeof(short)*(_inter.host_out_bit).size());
    err=hipMalloc((void **)&dev_rise_val, sizeof(float)*(_inter.host_rise_val).size());
    err=hipMalloc((void **)&dev_fall_val, sizeof(float)*(_inter.host_fall_val).size());
    err=hipMalloc((void **)&dev_functions, sizeof(short)*(_inter.host_functions).size());
    checkError(err,"cudamalloc error");
    
    err = hipMemcpy(dev_data_in_num_start, &(_inter.data_in_num_start)[0], sizeof(unsigned int)*(_inter.data_in_num_start).size(), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_data_out_num_start, &(_inter.data_out_num_start)[0], sizeof(unsigned int)*(_inter.data_out_num_start).size(), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_delay_start, &(_inter.delay_start)[0], sizeof(unsigned int)*(_inter.delay_start).size(), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_delay_width, &(_inter.delay_width)[0], sizeof(unsigned int)*(_inter.delay_width).size(), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_functions_start, &(_inter.functions_start)[0], sizeof(unsigned int)*(_inter.functions_start).size(), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_functions_width, &(_inter.functions_width)[0], sizeof(unsigned int)*(_inter.functions_width).size(), hipMemcpyHostToDevice);
    //err = hipMemcpy(dev_val_num_start, &val_num_start[0], sizeof(unsigned int)*val_num_start.size(), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_data_in_num, &(_inter.data_in_num)[0], sizeof(short)*(_inter.data_in_num).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_data_out_num, &(_inter.data_out_num)[0], sizeof(short)*(_inter.data_out_num).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_delay_val_num, &(_inter.delay_val_num)[0], sizeof(short)*(_inter.delay_val_num).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_functions_func_num, &(_inter.functions_func_num)[0], sizeof(short)*(_inter.functions_func_num).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_functions_val_num, &(_inter.functions_val_num)[0], sizeof(short)*(_inter.functions_val_num).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_functions, &(_inter.host_functions)[0], sizeof(short)*(_inter.host_functions).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_delay_edges, &(_inter.host_delay_edges)[0], sizeof(short)*(_inter.host_delay_edges).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_in_bit, &(_inter.host_in_bit)[0], sizeof(short)*(_inter.host_in_bit).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_out_bit, &(_inter.host_out_bit)[0], sizeof(short)*(_inter.host_out_bit).size(), hipMemcpyHostToDevice);            
    err=hipMemcpy(dev_rise_val, &(_inter.host_rise_val)[0], sizeof(float)*(_inter.host_rise_val).size(), hipMemcpyHostToDevice);
    err=hipMemcpy(dev_fall_val, &(_inter.host_fall_val)[0], sizeof(float)*(_inter.host_fall_val).size(), hipMemcpyHostToDevice);
    checkError(err,"cudamemcpy error");
            

    /*std::map<std::string, Instance> instances = _inter.p.get_instances();
    std::vector<string> _inst_name_vec = _inter.p.get_instance_names();*/
    std::map<std::string, std::string> assign_pairs = _inter.p.get_assign_pairs();
    std::map<std::string, TimedValues*> pinbitValues = _inter.p.get_pinbitValues();
    std::vector<std::map<std::string, std::vector<unsigned int> > > OutMaps = _inter.OutMaps;
    //std::vector<std::string> pin_bits = _inter.p.pin_bits;


    cout << "start parsing vcd file..." << endl;
    
    /** parse vcd & simulate **/
    VCDParser parser;// = new VCDParser();

    // replace pre-malloc at PreProcess.cpp
    parser.init(vcdFilePath, sim_end);
    // need use map at PreProcess.cpp
    std::vector<std::string> _pin_bits = _inter.p.pin_bits;
    for(auto it:_pin_bits){
        if(it.find("[") != std::string::npos){
            pin_bits_set.insert(it);
        }
    }
    //
    auto start_vcd = std::chrono::steady_clock::now();
    if(parser.parse(vcdFilePath, sim_start, sim_end))//, initial_net_map))
    {
        auto end_vcd1 = std::chrono::steady_clock::now();
        long duration_vcd1 = std::chrono::duration_cast<std::chrono::milliseconds>(end_vcd1 - start_vcd).count();
        cout << "total time of vcd parse(only parse): " << duration_vcd1 << "ms" << endl;
        parser.process();

        int tmpp = -1;
        long duration_gpu=0;
        long duration_pre=0;
        long duration_pro0=0;
        long duration_pro1=0;
        long duration_cuda=0;
        long duration_cuda2=0;
        long duration_out=0;
        long duration_out2 = 0;
        long duration_data = 0;
        long duration_malloc2 = 0;
        long duration_pro00 = 0;
        auto end_vcd = std::chrono::steady_clock::now();
        long duration_vcd = std::chrono::duration_cast<std::chrono::milliseconds>(end_vcd - start_vcd).count();
        cout << "total time of vcd parse: " << duration_vcd << "ms" << endl;
        //vcd signals
        VCDTimeUnit time_unit = TIME_PS;//parser.time_units;
        unsigned time_res = 1;//parser.time_resolution;
        //VCDScope *root = parser.root_scope;
        string root_name = "root";//root->name;


        
        /** Simulate **/
        cout << "start simulator..." << endl;
        auto start = std::chrono::steady_clock::now();
        
        //GPU cuda
        unsigned _size1 = _inter.levels.size();
        std::vector<unsigned int*> out_times(_size1); //unsigned int **out_times = (unsigned int **)malloc(sizeof(unsigned int*)*_size1);
        std::vector<short*> out_values(_size1);
        std::vector<unsigned int*> out_sizes(_size1);
        std::vector<unsigned int*> out_starts(_size1);
        std::unordered_map<std::string, std::vector<unsigned int> > vec_bit_times;
        std::unordered_map<std::string, std::vector<short> > vec_bit_values;

        for (unsigned i = 0; i < _size1; ++i)
        {
            //cout << "level " << i << endl;
            const int _cur_level = i;
            std::vector<int> cur_level = (_inter.levels)[i];
            const int inst_num = cur_level.size();
            
            int blocks_per_inst = 1;
            int n_blocks = inst_num;

            int height = inst_num;
            //data to GPU
            int _time_unit = time_unit;

            
            unsigned out_width=0;
            unsigned length_of_valid_width=0;
            unsigned out_width2=0;
            unsigned data_width=0;
            unsigned data_width_compress=0;
            unsigned input_width=0;
            int process_data_width = 0;

            
            unsigned int val_num_width=0;

            
            int functions_width=0;

            unsigned int times_width=0;
            
            int delay_width=0;

            //some flag
            //short *data_in_num = (short *)malloc(sizeof(short)*height);
            //short *data_out_num = (short *)malloc(sizeof(short)*height);


            unsigned int *input_datas_val_num = (unsigned int *)malloc(sizeof(unsigned int)*inst_num*MAX_IN_NUM);
            unsigned int _end = stoull(dumpoff_time_str);
            unsigned long long Length;
            if(_end > 1000000000)
                Length = _end * 2;
            else
                Length = _end * 25;
            //char *input_datas = new char[Length];
            short *input_datas = new short[Length / STATE_NUM + inst_num * MAX_IN_NUM * 2 / 3];
            
            //inst_num*N_THREADS_PER_BLOCK*N_TIMES_PER_THREAD*12);
            //unsigned int *input_times = (unsigned int *)malloc(sizeof(unsigned int)*200000000);//inst_num*N_THREADS_PER_BLOCK*N_TIMES_PER_THREAD*12);
            unsigned int input_datas_size = 0;
            //unsigned int *total_input_times = new unsigned int[Length];
            unsigned int *total_input_times = new unsigned int[Length];
            // for times compress
            //unsigned int *base_start = new unsigned int[Length / 10];
            //unsigned int *base_start_start = new unsigned int[inst_num * MAX_IN_NUM];
            //base_start[0] = 0;
            unsigned base_start_total = 0;
            unsigned max_base_start_idx = 0;
            unsigned base_start_start_idx = 0;

            //cout << inst_num << endl;
            unsigned int total_input_times_size = 0;
            std::vector<unsigned int> total_times_sizes(inst_num);
            std::vector<unsigned int> total_times_start(inst_num);
            std::vector<unsigned int> total_times_sizes_com(inst_num);
            std::vector<unsigned int> total_times_start_com(inst_num);

            //std::vector<int> input_start(inst_num);
            unsigned int *input_start = (unsigned int *)malloc(sizeof(unsigned int)*inst_num);
            unsigned int *output_start = (unsigned int *)malloc(sizeof(unsigned int)*inst_num);
            unsigned int *output_size = (unsigned int *)malloc(sizeof(unsigned int)*inst_num);
            unsigned int *valid_width_start = (unsigned int *)malloc(sizeof(unsigned int)*inst_num);
            unsigned int *valid_width_size = (unsigned int *)malloc(sizeof(unsigned int)*inst_num);
            //unsigned int *output_start2 = (unsigned int *)malloc(sizeof(unsigned int)*inst_num);
            //unsigned int *output_size2 = (unsigned int *)malloc(sizeof(unsigned int)*inst_num);
            if (output_size == NULL)
            {
                cout << "error: memory full!" << endl;
                exit(-1);
            }
            //int *val_num_start = (int *)malloc(sizeof(int)*inst_num);
            std::vector<unsigned int> val_num_start(inst_num);

            auto start_pro = std::chrono::steady_clock::now();
            unsigned int max_in_num = 0;
            //unsigned int max_single_num = 0;
            unsigned int times_start = 0;
            std::vector<unsigned int> merge_max_thread(MAX_IN_NUM);

            std::unordered_map<std::string, std::vector<unsigned long long> > temp_Result;
            //short temp_data;
            //int t_flag = 0;
            std::vector<string> name_vec;
            //std::vector<unsigned int> step_vec;
            std::vector<unsigned int*> time_vec;
            std::vector<short*> value_vec;
            std::vector<bool> is_init_vec;
            std::vector<short> offset_vec;
            std::vector<unsigned> step_start_vec;
            std::vector<unsigned> com_step_start_vec;
            std::vector<bool> cur_flag;
            name_vec.reserve(inst_num * MAX_IN_NUM * 2 / 3);
            time_vec.reserve(inst_num * MAX_IN_NUM * 2 / 3);
            value_vec.reserve(inst_num * MAX_IN_NUM * 2 / 3);
            is_init_vec.reserve(inst_num * MAX_IN_NUM * 2 / 3);
            offset_vec.reserve(inst_num * MAX_IN_NUM * 2 / 3);
            step_start_vec.reserve(inst_num * MAX_IN_NUM * 2 / 3);
            cur_flag.reserve(inst_num * MAX_IN_NUM * 2 / 3);
            unsigned long long step_sum = 0;
            unsigned int max_step = 0;
            for (int j = 0; j < inst_num; ++j)
            {
                Instance *cur_inst = _inter.find_inst(cur_level[j]);//&((_inter.p.get_instances())[(_inter.p.get_instance_names())[cur_level[j]]]);
                std::vector<string> cur_in = cur_inst->in_net;
                short data_in_num = cur_in.size();
                std::vector<string> cur_out = cur_inst->out_net;
                short data_out_num = cur_out.size();
                std::vector<int> in_net_from_id = cur_inst->in_net_from_id;
                std::vector<int> in_net_from_level = cur_inst->in_net_from_level;    
                std::vector<int> in_net_from_pos_at_level = cur_inst->in_net_from_pos_at_level; 
                unsigned int sum = 0;
                unsigned int sum_compress = 0;
                short offset = 0;
                for(unsigned it = 0; it < cur_in.size(); it++)
                {
                    unsigned int *pos = nullptr;
                    short *pos_val = nullptr;
                    string _name = cur_in[it];
                    name_vec.push_back(_name);
                    if(in_net_from_id[it] != -2 && Result.find(_name) == Result.end()){
                        cur_flag.push_back(false);
                        Result[_name].resize(3);
                        temp_Result[_name].resize(3);
                    }
                    else{
                        cur_flag.push_back(true);
                    }
                    unsigned int step;
                    if (in_net_from_id[it] == -1)
                    {
                        std::vector<std::string> in_net_from_info = cur_inst->in_net_from_info;
                        string in_name = in_net_from_info[it];

                        if((parser.sizes).find(in_name) != (parser.sizes).end()){
                            step = (parser.sizes)[in_name];
                            pos = &(*((parser.times)[in_name].begin()));
                            pos_val = &(*((parser.values)[in_name].begin()));
                            time_vec.push_back(pos);
                            value_vec.push_back(pos_val);
                            auto _pos = (parser.InMaps_bus).find(_name);
                            if(_pos != (parser.InMaps_bus).end()){
                                (parser.InMaps_bus).erase(_pos);
                            }
                        }
                        else{
                            string hash = (parser.hash_name_pair)[in_name];
                            unsigned _idx = (parser.hash_index_pair)[hash];
                            step = (parser.vec_times)[_idx].size();//(parser.vec_sizes)[_idx];//
                            pos = &(*((parser.vec_times)[_idx].begin()));
                            pos_val = &(*((parser.vec_values)[_idx].begin()));
                            time_vec.push_back(pos);
                            value_vec.push_back(pos_val);
                            auto _pos = (parser.InMaps_bit).find(_name);
                            if(_pos != (parser.InMaps_bit).end()){
                                (parser.InMaps_bit).erase(_pos);
                            }
                        }
                        
                        step_start_vec.push_back(sum+data_width);
                        com_step_start_vec.push_back(sum_compress + data_width_compress);
                        max_step = max(step, max_step);
                        sum += step;
                        sum_compress += (step + STATE_NUM - 1) / STATE_NUM;
                        //step_vec.push_back(step);
                        is_init_vec.push_back(true);
                        offset_vec.push_back(0);
                        /*
                        pos = &(*((parser.times)[in_name].begin()));
                        pos_val = &(*((parser.values)[in_name].begin()));
                        time_vec.push_back(pos);
                        value_vec.push_back(pos_val);
                        if((parser.sizes).find(in_name) != (parser.sizes).end()){
                            step = (parser.sizes)[in_name];
                        }else{
                            step = (parser.times)[in_name].size();
                        }
                        step_start_vec.push_back(sum+data_width);
                        sum += step;
                        //step_vec.push_back(step);
                        is_init_vec.push_back(true);
                        offset_vec.push_back(0);*/
                    }
                    else if (in_net_from_id[it] >= 0)
                    {
                        std::vector<int> in_net_from_out_order = cur_inst->in_net_from_out_order;
                        int _level = in_net_from_level[it];
                        int _level_pos = in_net_from_pos_at_level[it];  
                        auto _pos = OutMaps[_level].find(_name);
                        if(_pos != OutMaps[_level].end()){
                            OutMaps[_level].erase(_pos);
                        } 
                        
                        unsigned tmp_step = *(out_sizes[_level] + _level_pos * MAX_OUT_NUM);
                        step = *(out_sizes[_level] + _level_pos * MAX_OUT_NUM + in_net_from_out_order[it]);
                        unsigned int _out_start = *(out_starts[_level] + _level_pos) + in_net_from_out_order[it] * tmp_step;
                        pos = out_times[_level] + _out_start;
                        pos_val = out_values[_level] + _out_start / STATE_NUM;
                        time_vec.push_back(pos);
                        value_vec.push_back(pos_val);
                        offset = _out_start % STATE_NUM;
                        step_start_vec.push_back(sum+data_width);
                        com_step_start_vec.push_back(sum_compress + data_width_compress);
                        max_step = max(step, max_step);
                        sum += step;
                        sum_compress += (step + STATE_NUM - 1) / STATE_NUM;
                        //step_vec.push_back(step);
                        is_init_vec.push_back(false);
                        offset_vec.push_back(offset);
                    }
                    else if (in_net_from_id[it] == -2)
                    {
                        std::vector<std::string> in_net_from_info = cur_inst->in_net_from_info;
                        string in_name = in_net_from_info[it];

                        TimedValues* te = pinbitValues[in_name];
                        auto tev = te->begin();
                        step = te->size();
                        unsigned int* _tvs = (unsigned int *)malloc(sizeof(unsigned int)*step);
                        short* _tvs_val = (short *)malloc(sizeof(short)*step);
                        for (unsigned k = 0; k < step; ++k)
                        {
                            TimedValue _tv = *(tev+k);
                            _tvs[k] = _tv.t;
                            _tvs_val[k] = _tv.value;
                        }
                        pos = _tvs;
                        pos_val = _tvs_val;
                        time_vec.push_back(pos);
                        value_vec.push_back(pos_val);
                        step_start_vec.push_back(sum+data_width);
                        com_step_start_vec.push_back(sum_compress + data_width_compress);
                        max_step = max(step, max_step);
                        sum += step;
                        sum_compress += (step + STATE_NUM - 1) / STATE_NUM;
                        //step_vec.push_back(step);
                        is_init_vec.push_back(true);
                        offset_vec.push_back(0);
                    }
                    else
                    {
                        cout << "error." << endl;
                        exit(-1);
                    }
                    step_sum += step;
                    input_datas_val_num[it+val_num_width] = step;
                    merge_max_thread[it] = max(merge_max_thread[it], sum);
                } 
                //
                //int tmp_size = sum;//tmp_times.size();
                total_times_sizes[j] = sum;//tmp_size;
                total_times_start[j] = data_width;
                //total_times_sizes_com[j] = sum_compress;//tmp_size;
                //total_times_start_com[j] = data_width_compress;
                input_start[j] = input_width;

                
                int _blocks_per_inst = (sum + N_THREADS_PER_BLOCK * N_TIMES_PER_THREAD - 1) / (N_THREADS_PER_BLOCK * N_TIMES_PER_THREAD);
                if (_blocks_per_inst*inst_num/MAX_BLOCKS > MAX_BLOCKS)
                {
                    _blocks_per_inst = (MAX_BLOCKS + inst_num - 1) / inst_num * MAX_BLOCKS;
                }

                unsigned int _out_width = sum;
                unsigned int _length_of_valid_width = (sum + N_TIMES_PER_THREAD - 1) / N_TIMES_PER_THREAD;
                output_start[j] = out_width;
                valid_width_start[j] = length_of_valid_width;
                output_size[j] = _out_width;
                valid_width_size[j] = _length_of_valid_width;
                out_width += _out_width*data_out_num;
                length_of_valid_width += _length_of_valid_width * data_out_num;

                blocks_per_inst = max(blocks_per_inst, _blocks_per_inst);

                val_num_start[j] = val_num_width;
                unsigned int _val_num_width = data_in_num;
                max_in_num = max(max_in_num, _val_num_width);
                val_num_width += _val_num_width;
                data_width += sum;//tmp_size*data_in_num[j];
                data_width_compress += sum_compress;
                input_width += sum*data_in_num;                
            }
            //data_width_compress = (data_width + STATE_NUM - 1 ) / STATE_NUM;
            auto end_pro000 = std::chrono::steady_clock::now();
            duration_pro00 += std::chrono::duration_cast<std::chrono::microseconds>(end_pro000 - start_pro).count();

            // CPU multiple threads
            unsigned size = name_vec.size();
            unsigned max_cores = (size > thread::hardware_concurrency()*6) ? (thread::hardware_concurrency()*6) : size;
            vector<thread> threads(max_cores);
            unsigned left = 0;
            //unsigned num_every_thread = (size + max_cores - 1) / max_cores;
            unsigned long long num_every_thread = step_sum / max_cores;
            vector<unsigned> right_vec(max_cores);
            for(unsigned t = 0; t < max_cores; ++t){
                unsigned ll;
                unsigned long long _size = 0;
                for(ll = left; ll < size && _size < num_every_thread; ++ll){
                    _size += input_datas_val_num[ll];
                }
                right_vec[t] = ll;
                left = ll;
            }
            left = 0;
            for(unsigned t = 0; t < max_cores; ++t){
                unsigned right = right_vec[t];//left + num_every_thread;
                //cout << "start thread" << endl;
                /*threads[t] = thread(processCore, left, right, inst_num, std::ref(_inter), std::ref(parser), sim_start, 
                    sim_end, total_input_times, input_datas, std::ref(cur_level), std::ref(out_starts), std::ref(out_sizes),
                    std::ref(out_times), std::ref(out_values), std::ref(pinbitValues),
                    std::ref(total_times_start), std::ref(cur_flag), std::ref(temp_Result));*/
                threads[t] = thread(processCore, left, right, size, sim_start, sim_end, 
                    total_input_times, input_datas, std::ref(name_vec), std::ref(time_vec), std::ref(value_vec),
                    input_datas_val_num ,std::ref(is_init_vec), std::ref(offset_vec), 
                    std::ref(step_start_vec), std::ref(com_step_start_vec), std::ref(cur_flag), std::ref(temp_Result));
                left = right;
            }
            for(auto &t:threads){
                t.join();
            }

            // Done multiple-thread
            
            

            int n_block_num2 = blocks_per_inst;//(inst_num * blocks_per_inst + MAX_BLOCKS - 1) / MAX_BLOCKS;
            int n_block_num1 = inst_num;//(inst_num * blocks_per_inst + n_block_num2 - 1) / n_block_num2;
            dim3 n_grid(n_block_num1, n_block_num2, 1);
            //n_blocks = inst_num * blocks_per_inst;
            
            //cout << "blocks_per_inst: " << blocks_per_inst << endl;
            //out_width = out_width * max_out;
            
            //sort(total_input_times->begin(),total_input_times->end());  
            //total_input_times->erase(unique(total_input_times->begin(),total_input_times->end()),total_input_times->end());
            //cout << "total_input_times size: " << total_input_times->size() << endl;
            //cout << "data_width size: " << data_width << endl;
            //cout << "data_width_compress size: " << data_width_compress << endl;
            //cout << "max_in size: " << max_in_num << endl;
            //out_width = total_input_times->size() * max_out;
            auto end_pro = std::chrono::steady_clock::now();
            long duration_pro = std::chrono::duration_cast<std::chrono::milliseconds>(end_pro - start_pro).count();
            //cout << "total time of proccess: " << duration_pro << "ms" << endl;
            //cout << "total time of proccess(us): " << duration_pro00 << "us" << endl;
            duration_pre += duration_pro;
            
            auto start_data = std::chrono::steady_clock::now();


            unsigned int *dev_total_times_sizes;
            err=hipMalloc((void **)&dev_total_times_sizes, sizeof(unsigned int)*height);
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(dev_total_times_sizes) on GPU is failed, return value is %d\n", err);
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            err = hipMemcpy(dev_total_times_sizes, &total_times_sizes[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);
            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_total_times_sizes) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            unsigned int *dev_total_times_start;
            err=hipMalloc((void **)&dev_total_times_start, sizeof(unsigned int)*height);
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(dev_total_times_start) on GPU is failed, return value is %d\n", err);
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            err = hipMemcpy(dev_total_times_start, &total_times_start[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);
            //unsigned int *dev_total_times_sizes_com;
            //err=hipMalloc((void **)&dev_total_times_sizes_com, sizeof(unsigned int)*height);
            //if(err!=hipSuccess)
            //{
            //    printf("the hipMalloc(dev_total_times_sizes) on GPU is failed, return value is %d\n", err);
            //    cout << hipGetErrorString(err) << endl;
            //   exit(-1);
            //}
            //err = hipMemcpy(dev_total_times_sizes_com, &total_times_sizes_com[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);
            //if(err!=hipSuccess)
            //{
            //    printf("the hipMemcpy(dev_total_times_sizes) on GPU is failed\n");
            //    cout << hipGetErrorString(err) << endl;
            //    exit(-1);
            //}
            /*unsigned int *dev_total_times_start_com;
            err=hipMalloc((void **)&dev_total_times_start_com, sizeof(unsigned int)*height);
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(dev_total_times_start) on GPU is failed, return value is %d\n", err);
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            err = hipMemcpy(dev_total_times_start_com, &total_times_start_com[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);*/
            //Decompress GPU
            short *dev_in_datas;
            //char *dev_in_data_compressed;
            short *dev_in_data_compressed;
            unsigned int *dev_step_start_vec;
            unsigned int *dev_com_step_start_vec;
            unsigned int *dev_data_val_num;
            unsigned com_size = step_start_vec.size();
            err=hipMalloc((void **)&dev_in_datas, sizeof(short)*data_width);
            err=hipMalloc((void **)&dev_in_data_compressed, sizeof(short)*data_width_compress);
            err=hipMalloc((void **)&dev_step_start_vec, sizeof(unsigned int)*com_size);
            err=hipMalloc((void **)&dev_com_step_start_vec, sizeof(unsigned int)*com_size);
            err=hipMalloc((void **)&dev_data_val_num, sizeof(unsigned int)*val_num_width);
            err = hipMemcpy(dev_in_data_compressed, &input_datas[0], sizeof(short)*data_width_compress, hipMemcpyHostToDevice);
            err=hipMemcpy(dev_step_start_vec, &step_start_vec[0], sizeof(unsigned int)*com_size, hipMemcpyHostToDevice);
            err=hipMemcpy(dev_com_step_start_vec, &com_step_start_vec[0], sizeof(unsigned int)*com_size, hipMemcpyHostToDevice);
            err=hipMemcpy(dev_data_val_num, &input_datas_val_num[0], sizeof(unsigned int)*val_num_width, hipMemcpyHostToDevice);

            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_in_data_compressed) on GPU is failed, return value is %d\n", err);
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            //int _n_block_num2 = (data_width / N_THREADS_PER_BLOCK > MAX_BLOCKS) ? MAX_BLOCKS : 1;
            //int _n_block_num1 = (data_width / N_THREADS_PER_BLOCK > MAX_BLOCKS) ? ((data_width + N_THREADS_PER_BLOCK*MAX_BLOCKS - 1)/(N_THREADS_PER_BLOCK*MAX_BLOCKS)) : ((data_width + N_THREADS_PER_BLOCK - 1)/N_THREADS_PER_BLOCK);
            unsigned int blocks_per_signal = (((max_step + STATE_NUM - 1) / STATE_NUM) + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK;
            int _n_block_num2 = (com_size * blocks_per_signal > MAX_BLOCKS) ? MAX_BLOCKS : 1;
            int _n_block_num1 = (com_size * blocks_per_signal > MAX_BLOCKS) ? ((com_size * blocks_per_signal + MAX_BLOCKS - 1)/(MAX_BLOCKS)) : (com_size * blocks_per_signal);
            //cout << "_n_grid:" << _n_block_num1 << ", " << _n_block_num2 << endl;
            dim3 _n_grid(_n_block_num1, _n_block_num2, 1);
            DecompressGPU<<<_n_grid, N_THREADS_PER_BLOCK>>>(dev_in_datas, dev_in_data_compressed, dev_step_start_vec, 
                dev_com_step_start_vec, dev_data_val_num, blocks_per_signal, com_size);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                cout << "DecompressGPU kenerl error: " << hipGetErrorString(err) << endl;
            } 
            hipFree(dev_in_data_compressed);
            hipFree(dev_step_start_vec);
            hipFree(dev_com_step_start_vec);

            // Done Decompress
            
            unsigned int *dev_times;
            total_input_times_size = data_width;
            err=hipMalloc((void **)&dev_times, sizeof(unsigned int)*total_input_times_size);
            err=hipMemcpy(dev_times, &(total_input_times[0]), sizeof(unsigned int)*total_input_times_size, hipMemcpyHostToDevice);
            
            unsigned int *dev_input_start;
            err=hipMalloc((void **)&dev_input_start, sizeof(unsigned int)*height);
            err = hipMemcpy(dev_input_start, &input_start[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);


            short *dev_datas;
            err = hipMalloc((void **)&dev_datas, sizeof(short)*input_width);
            
            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_total_times_start) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }

            /*if(_cur_level == 4){
                ofstream outFile("test.dat", ios::out | ios::binary);
                outFile.write((char*)&total_input_times[0], sizeof(unsigned int)*total_input_times_size);
                outFile.close();
            }*/

            
            
            unsigned int *dev_val_num_start;
            err=hipMalloc((void **)&dev_val_num_start, sizeof(unsigned int)*height);
            
            err = hipMemcpy(dev_val_num_start, &val_num_start[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);
            
            if(data_width > 390000000){
                
                unsigned data_width1 = total_times_start[(inst_num + 1) / 2];
                unsigned data_width2 = data_width - total_times_start[(inst_num + 1) / 2];
                unsigned input_width1 = input_start[(inst_num + 1) / 2];
                unsigned input_width2 = input_width - input_start[(inst_num + 1) / 2];
                //err = hipMalloc((void **)&dev_in_datas, sizeof(short)*data_width1);
                if(err!=hipSuccess)
                {
                    printf("the hipMalloc(in data) on GPU is failed");
                    exit(-1);
                }

                // GPU Merge sort
                unsigned int *dev_temp_times;
                err=hipMalloc((void **)&dev_temp_times, sizeof(unsigned int)*data_width1);
                //err=hipMemset(dev_temp_times, 0, sizeof(unsigned int)*total_input_times_size);
            
                short *dev_temp_datas;
            
                err = hipMalloc((void **)&dev_temp_datas, sizeof(short)*input_width1);
    
                //err=hipMemcpy(dev_in_datas, &input_datas[0], sizeof(short)*data_width1, hipMemcpyHostToDevice);
                
                auto start_data2 = std::chrono::steady_clock::now();
                for (unsigned j = 1; j < max_in_num; j++)
                {
                    //cout << "CUDA merge." << j << endl;
                    unsigned loop_num = 1;
                    int block_num1, block_num2, block_num;
                    int _blocks_per_inst;
                    unsigned int _merge_max_thread = max(merge_max_thread[0], merge_max_thread[j]);
                    if (inst_num / 2 *((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK) / _MAX_BLOCKS > _MAX_BLOCKS)
                    {
                        //cout << "CUDA merge multiple." << endl;
                        unsigned num = inst_num;
                        unsigned temp = inst_num*((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK);
                        while(temp > _MAX_BLOCKS)
                        {
                            num = (inst_num + loop_num) / (loop_num + 1);
                            temp = num*((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK);
                            loop_num++;
                        }
                        int every_loop_num = (inst_num + loop_num - 1) / loop_num;
                        int cur_inst_num = inst_num;
                        int inst_id_start = 0;
                        while(loop_num > 0)
                        {
                            if (loop_num == 1)
                            {
                                every_loop_num = cur_inst_num;
                            }
                            _blocks_per_inst = (_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK;
                            block_num = every_loop_num*_blocks_per_inst;
                            PreCudaMerge<<<block_num, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, dev_in_datas, _blocks_per_inst, _cur_level, inst_id_start, 0, (inst_num + 1) / 2,
                                dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                                dev_temp_times, dev_temp_datas);
                            hipDeviceSynchronize();
                            err = hipGetLastError();
                            if (err != hipSuccess)
                            {
                                cout << "cuda merge kenerl error: " << hipGetErrorString(err) << endl;
                            }  
                            CudaMerge<<<block_num, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, _blocks_per_inst, _cur_level, inst_id_start, 0, (inst_num + 1) / 2,
                                dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                                dev_temp_times, dev_temp_datas);
                            hipDeviceSynchronize();
                            err = hipGetLastError();
                            if (err != hipSuccess)
                            {
                                cout << "cuda merge2 kenerl error: " << hipGetErrorString(err) << endl;
                            }  
                            loop_num--;
                            cur_inst_num -= every_loop_num;
                            inst_id_start += every_loop_num;
                        }
                    }
                    else{
                        _blocks_per_inst = (_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK;
                        block_num2 = (inst_num * _blocks_per_inst + _MAX_BLOCKS - 1) / _MAX_BLOCKS;
                        block_num1 = (inst_num * _blocks_per_inst + block_num2 - 1) / block_num2;
                        dim3 grid(block_num1, block_num2, 1);
                        int inst_id_start = 0;
                        PreCudaMerge<<<grid, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, dev_in_datas, _blocks_per_inst, _cur_level, inst_id_start, 0, (inst_num + 1) / 2,
                            dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                            dev_temp_times, dev_temp_datas);
                        hipDeviceSynchronize();
                        err = hipGetLastError();
                        if (err != hipSuccess)
                        {
                            cout << "cuda merge kenerl error: " << hipGetErrorString(err) << endl;
                        }     
                        CudaMerge<<<grid, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, _blocks_per_inst, _cur_level, inst_id_start, 0, (inst_num + 1) / 2,
                            dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                            dev_temp_times, dev_temp_datas);
                        hipDeviceSynchronize();
                        err = hipGetLastError();
                        if (err != hipSuccess)
                        {
                            cout << "cuda merge2 kenerl error: " << hipGetErrorString(err) << endl;
                        }  
                    }
                }
                auto end_data2 = std::chrono::steady_clock::now();
                long duration_data2 = std::chrono::duration_cast<std::chrono::milliseconds>(end_data2 - start_data2).count();
                duration_malloc2 += duration_data2;
                hipFree(dev_temp_times);
                hipFree(dev_temp_datas);
                //hipFree(dev_in_datas);
                // Next
                //err = hipMalloc((void **)&dev_in_datas, sizeof(short)*data_width2);
                if(err!=hipSuccess)
                {
                    printf("the hipMalloc(in data) on GPU is failed");
                    exit(-1);
                }
                // GPU Merge sort
                err=hipMalloc((void **)&dev_temp_times, sizeof(unsigned int)*data_width2);
                err = hipMalloc((void **)&dev_temp_datas, sizeof(short)*input_width2);
                //err=hipMemcpy(dev_in_datas, input_datas+data_width1, sizeof(short)*data_width2, hipMemcpyHostToDevice);
                if(err!=hipSuccess)
                {
                    printf("the hipMemcpy(dev_in_datas) on GPU is failed\n");
                    cout << hipGetErrorString(err) << endl;
                    exit(-1);
                }

                start_data2 = std::chrono::steady_clock::now();
                for (unsigned j = 1; j < max_in_num; j++)
                {
                    unsigned loop_num = 1;
                    int block_num1, block_num2, block_num;
                    int _blocks_per_inst;
                    unsigned int _merge_max_thread = max(merge_max_thread[0], merge_max_thread[j]);
                    if (inst_num / 2 *((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK) / _MAX_BLOCKS > _MAX_BLOCKS)
                    {
                        unsigned num = inst_num;
                        unsigned temp = inst_num*((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK);
                        while(temp > _MAX_BLOCKS)
                        {
                            num = (inst_num + loop_num) / (loop_num + 1);
                            temp = num*((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK);
                            loop_num++;
                        }
                        int every_loop_num = (inst_num + loop_num - 1) / loop_num;
                        int cur_inst_num = inst_num;
                        int inst_id_start = 0;
                        while(loop_num > 0)
                        {
                            if (loop_num == 1)
                            {
                                every_loop_num = cur_inst_num;
                            }
                            _blocks_per_inst = (_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK;
                            block_num = every_loop_num*_blocks_per_inst;
                            PreCudaMerge<<<block_num, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, dev_in_datas, _blocks_per_inst, _cur_level, inst_id_start, (inst_num + 1) / 2, inst_num, 
                                dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                                dev_temp_times, dev_temp_datas);
                            hipDeviceSynchronize();
                            err = hipGetLastError();
                            if (err != hipSuccess)
                            {
                                cout << "cuda merge kenerl error: " << hipGetErrorString(err) << endl;
                            }  
                            CudaMerge<<<block_num, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, _blocks_per_inst, _cur_level, inst_id_start, (inst_num + 1) / 2, inst_num, 
                                dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                                dev_temp_times, dev_temp_datas);
                            hipDeviceSynchronize();
                            err = hipGetLastError();
                            if (err != hipSuccess)
                            {
                                cout << "cuda merge2 kenerl error: " << hipGetErrorString(err) << endl;
                            }  
                            loop_num--;
                            cur_inst_num -= every_loop_num;
                            inst_id_start += every_loop_num;
                        }
                    }
                    else{
                        _blocks_per_inst = (_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK;
                        block_num2 = (inst_num * _blocks_per_inst + _MAX_BLOCKS - 1) / _MAX_BLOCKS;
                        block_num1 = (inst_num * _blocks_per_inst + block_num2 - 1) / block_num2;
                        dim3 grid(block_num1, block_num2, 1);
                        //cout << "block_num: " << block_num1 * block_num2 << ", blocks_per_inst: " << _blocks_per_inst << endl;
                        int inst_id_start = 0;
                        PreCudaMerge<<<grid, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, dev_in_datas, _blocks_per_inst, _cur_level, inst_id_start, (inst_num + 1) / 2, inst_num, 
                            dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                            dev_temp_times, dev_temp_datas);
                        hipDeviceSynchronize();
                        err = hipGetLastError();
                        if (err != hipSuccess)
                        {
                            cout << "cuda merge kenerl error: " << hipGetErrorString(err) << endl;
                        }     
                        CudaMerge<<<grid, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, _blocks_per_inst, _cur_level, inst_id_start, (inst_num + 1) / 2, inst_num, 
                            dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                            dev_temp_times, dev_temp_datas);
                        hipDeviceSynchronize();
                        err = hipGetLastError();
                        if (err != hipSuccess)
                        {
                            cout << "cuda merge2 kenerl error: " << hipGetErrorString(err) << endl;
                        }  
                    }
                }
                end_data2 = std::chrono::steady_clock::now();
                duration_data2 = std::chrono::duration_cast<std::chrono::milliseconds>(end_data2 - start_data2).count();
                duration_malloc2 += duration_data2;
            //
                hipFree(dev_temp_times);
                hipFree(dev_temp_datas);
                hipFree(dev_in_datas);
            }
            else{
                //short *dev_in_datas;
                //err = hipMalloc((void **)&dev_in_datas, sizeof(short)*data_width);
                if(err!=hipSuccess)
                {
                    printf("the hipMalloc(in data) on GPU is failed");
                    exit(-1);
                }
                // GPU Merge sort
                unsigned int *dev_temp_times;
                short *dev_temp_datas;
                err=hipMalloc((void **)&dev_temp_times, sizeof(unsigned int)*data_width);
                err = hipMalloc((void **)&dev_temp_datas, sizeof(short)*input_width);
                //err=hipMemcpy(dev_in_datas, input_datas, sizeof(short)*data_width, hipMemcpyHostToDevice);
                if(err!=hipSuccess)
                {
                    printf("the hipMemcpy(dev_in_datas) on GPU is failed\n");
                    cout << hipGetErrorString(err) << endl;
                    exit(-1);
                }

                auto start_data2 = std::chrono::steady_clock::now();
                for (unsigned j = 1; j < max_in_num; j++)
                {
                    unsigned loop_num = 1;
                    int block_num1, block_num2, block_num;
                    int _blocks_per_inst;
                    unsigned int _merge_max_thread = max(merge_max_thread[0], merge_max_thread[j]);
                    if (inst_num / 2 *((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK) / _MAX_BLOCKS > _MAX_BLOCKS)
                    {
                        unsigned num = inst_num;
                        unsigned temp = inst_num*((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK);
                        while(temp > _MAX_BLOCKS)
                        {
                            num = (inst_num + loop_num) / (loop_num + 1);
                            temp = num*((_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK);
                            loop_num++;
                        }
                        int every_loop_num = (inst_num + loop_num - 1) / loop_num;
                        int cur_inst_num = inst_num;
                        int inst_id_start = 0;
                        while(loop_num > 0)
                        {
                            if (loop_num == 1)
                            {
                                every_loop_num = cur_inst_num;
                            }
                            _blocks_per_inst = (_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK;
                            block_num = every_loop_num*_blocks_per_inst;
                            PreCudaMerge<<<block_num, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, dev_in_datas, _blocks_per_inst, _cur_level, inst_id_start, 0, inst_num, 
                                dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                                dev_temp_times, dev_temp_datas);
                            hipDeviceSynchronize();
                            err = hipGetLastError();
                            if (err != hipSuccess)
                            {
                                cout << "cuda merge kenerl error: " << hipGetErrorString(err) << endl;
                            }  
                            CudaMerge<<<block_num, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, _blocks_per_inst, _cur_level, inst_id_start, 0, inst_num, 
                                dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                                dev_temp_times, dev_temp_datas);
                            hipDeviceSynchronize();
                            err = hipGetLastError();
                            if (err != hipSuccess)
                            {
                                cout << "cuda merge2 kenerl error: " << hipGetErrorString(err) << endl;
                            }  
                            loop_num--;
                            cur_inst_num -= every_loop_num;
                            inst_id_start += every_loop_num;
                        }
                    }
                    else{
                        _blocks_per_inst = (_merge_max_thread + _N_THREADS_PER_BLOCK - 1) / _N_THREADS_PER_BLOCK;
                        block_num2 = (inst_num * _blocks_per_inst + _MAX_BLOCKS - 1) / _MAX_BLOCKS;
                        block_num1 = (inst_num * _blocks_per_inst + block_num2 - 1) / block_num2;
                        dim3 grid(block_num1, block_num2, 1);
                        int inst_id_start = 0;
                        PreCudaMerge<<<grid, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, dev_in_datas, _blocks_per_inst, _cur_level, inst_id_start, 0, inst_num, 
                            dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                            dev_temp_times, dev_temp_datas);
                        hipDeviceSynchronize();
                        err = hipGetLastError();
                        if (err != hipSuccess)
                        {
                            cout << "cuda merge kenerl error: " << hipGetErrorString(err) << endl;
                        }     
                        CudaMerge<<<grid, _N_THREADS_PER_BLOCK>>>(j, dev_times, dev_datas, _blocks_per_inst, _cur_level, inst_id_start, 0, inst_num, 
                            dev_total_times_start, dev_input_start, dev_data_val_num, dev_val_num_start, dev_data_in_num_start, dev_data_in_num,
                            dev_temp_times, dev_temp_datas);
                        hipDeviceSynchronize();
                        err = hipGetLastError();
                        if (err != hipSuccess)
                        {
                            cout << "cuda merge2 kenerl error: " << hipGetErrorString(err) << endl;
                        }  
                    }
                }
                auto end_data2 = std::chrono::steady_clock::now();
                long duration_data2 = std::chrono::duration_cast<std::chrono::milliseconds>(end_data2 - start_data2).count();
                duration_malloc2 += duration_data2;
cout << "merge runtime (ms):" << duration_data2 << "--max_in_num:" << max_in_num << endl;
                hipFree(dev_temp_times);
                hipFree(dev_temp_datas);
                hipFree(dev_in_datas);
            }
            hipFree(dev_data_val_num);
            hipFree(dev_val_num_start);
            delete[] total_input_times;
            delete[] input_datas;
            free(input_datas_val_num);


            auto start_malloc = std::chrono::steady_clock::now();

            unsigned int *dev_output_start;
            err=hipMalloc((void **)&dev_output_start, sizeof(unsigned int)*height);
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(dev_output_start) on GPU is failed, return value is %d\n", err);
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }

            unsigned int *dev_output_size;
            err=hipMalloc((void **)&dev_output_size, sizeof(unsigned int)*height);
            //err=hipMalloc((void **)&dev_output_size2, sizeof(unsigned int)*height);
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(dev_output_size) on GPU is failed, return value is %d\n", err);
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            
            
            unsigned int *dev_out_times;
            short *dev_out_datas;  // 2D
            short *dev_out_splited_width;
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(dev_input_start) on GPU is failed, return value is %d\n", err);
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
        

        
            err=hipMalloc((void **)&dev_out_splited_width, sizeof(short)*height);
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(dev_out_splited_width) on GPU is failed, return value is %d\n", err);
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            
            
            err=hipMalloc((void **)&dev_out_times, sizeof(unsigned int)*out_width);
            
            //cout << "out_width: " << out_width << ", height: " << height << endl;
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(dev_out_times) on GPU is failed");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            
            
            
            err = hipMalloc((void **)&dev_out_datas, sizeof(short)*out_width);
            
            if(err!=hipSuccess)
            {
                printf("the hipMalloc(out data) on GPU is failed");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            //printf("SUCCESS\n");
            auto end_malloc = std::chrono::steady_clock::now();
            long duration_malloc = std::chrono::duration_cast<std::chrono::milliseconds>(end_malloc - start_malloc).count();
            //cout << "total time of simulating: " << duration_malloc << "ms" << endl;
            //printf("start copy memory\n");
            auto start_cpy = std::chrono::steady_clock::now();
            
            err = hipMemset(dev_out_splited_width, tmpp, sizeof(short)*height);
            //err = hipMemcpy(dev_out_splited_width, &host_out_splited_width[0], sizeof(int)*height, hipMemcpyHostToDevice);
            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_out_splited_width) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            err = hipMemcpy(dev_output_start, &output_start[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);
            //err = hipMemcpy(dev_output_start2, &output_start2[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);
            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_output_start) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            err = hipMemcpy(dev_output_size, &output_size[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);
            //err = hipMemcpy(dev_output_size2, &output_size2[0], sizeof(unsigned int)*height, hipMemcpyHostToDevice);
            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_output_size) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            
            
            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_input_start) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            
            

            
            err = hipMemset(dev_out_datas, tmpp, sizeof(short)*out_width);
            //err=hipMemcpy(dev_out_datas, host_out_data, sizeof(int)*out_width*height, hipMemcpyHostToDevice);  
            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_out_datas) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }          
            
            unsigned int ttmmpp = 0;
            err = hipMemset(dev_out_times, ttmmpp, sizeof(unsigned int)*out_width);
            //err=hipMemcpy(dev_out_times, out_times, sizeof(int)*out_width*height, hipMemcpyHostToDevice);
            if(err!=hipSuccess)
            {
                printf("the hipMemcpy(dev_out_times) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            
            unsigned int *dev_valid_width;
            unsigned int *dev_valid_width_start;
            unsigned int *dev_valid_width_size;
            unsigned int ch_init = 0;
            err = hipMalloc((void**)&dev_valid_width, sizeof(unsigned int) * length_of_valid_width);
            err = hipMalloc((void**)&dev_valid_width_start, sizeof(unsigned int) * inst_num);
            err = hipMalloc((void**)&dev_valid_width_size, sizeof(unsigned int) * inst_num);
            err = hipMemset(dev_valid_width, ch_init, sizeof(unsigned int)*length_of_valid_width);
            err = hipMemcpy(dev_valid_width_start, &valid_width_start[0], sizeof(unsigned int)*inst_num, hipMemcpyHostToDevice);
            err = hipMemcpy(dev_valid_width_size, &valid_width_size[0], sizeof(unsigned int)*inst_num, hipMemcpyHostToDevice);

            auto end_cpy = std::chrono::steady_clock::now();
            long duration_cpy = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpy - start_cpy).count();
            auto end_data = std::chrono::steady_clock::now();
            //cout << "total time of memcpy: " << duration_cpy << "ms" << endl;
            duration_cuda += (duration_malloc + duration_cpy);
            duration_data += std::chrono::duration_cast<std::chrono::milliseconds>(end_data - start_data).count();
            // GPU parallel
            //printf("Start simulate with GPU\n");
            //int num_thread = N_THREADS_PER_BLOCK;
            auto start_gpu = std::chrono::steady_clock::now();
            SimulateCuda<<<n_grid,N_THREADS_PER_BLOCK>>>(_cur_level,blocks_per_inst, dev_datas,dev_out_datas,dev_times,dev_out_times,//dev_data_val_num,dev_in_times,
                dev_functions,dev_delay_edges,dev_in_bit,dev_out_bit,dev_rise_val,dev_fall_val,
                _time_unit,dev_total_times_start,dev_total_times_sizes,dev_data_in_num,dev_data_out_num,dev_delay_val_num,dev_functions_func_num,dev_functions_val_num,dev_input_start,//dev_val_num_start,
                //functions_width,delay_width,
                dev_out_splited_width,dev_output_start,dev_output_size, dev_data_in_num_start,
                dev_data_out_num_start,dev_delay_start,dev_delay_width,dev_functions_start,dev_functions_width,
                dev_valid_width, dev_valid_width_start, dev_valid_width_size);
            if(_cur_level == 0){
                DumpSaif(outf, time_unit, time_res, root_name, sim_start, sim_end);
                //DumpSaif(outf, parser.in_Result);
            }
            DumpSaif(outf, temp_Result);
            //else{
                //DumpSaif(outf, temp_Result);
            if(_cur_level == _size1 - 1){
                DumpSaif(outf, _inter.p.Result, sim_start, sim_end);
            }
            //}
            hipDeviceSynchronize();
            auto end_gpu = std::chrono::steady_clock::now();
            duration_gpu += std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count();
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                cout << "kenerl error: " << hipGetErrorString(err) << endl;
            }
            //unsigned int *host_valid_width1 = (unsigned int*)malloc(sizeof(unsigned int) * length_of_valid_width);
            //err = hipMemcpy(host_valid_width1, dev_valid_width, sizeof(unsigned int)*length_of_valid_width, hipMemcpyDeviceToHost);
            

            free(input_start);
            hipFree(dev_total_times_start);//
            hipFree(dev_total_times_sizes);//
            hipFree(dev_input_start);//
            err=hipFree(dev_times);//
            if (err != hipSuccess)
            {
                printf("the cudafree(dev_times) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            err=hipFree(dev_datas);//
            //err=hipFree(dev_in_datas);//
            if (err != hipSuccess)
            {
                printf("the cudafree(dev_datas) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }


            // GPU merge output data
            //printf("Start copy memory to host\n");
            short *dev_valid_width_flag;
            err = hipMalloc((void**)&dev_valid_width_flag, sizeof(short) * length_of_valid_width);
            err = hipMemset(dev_valid_width_flag, 0, sizeof(short)*length_of_valid_width);
            auto start_out = std::chrono::steady_clock::now();
            short *out_data = (short *)malloc(sizeof(short)*out_width);
            //short *out_data = (short *)malloc(sizeof(short)*ceil(out_width / STATE_NUM));
            unsigned int *out_time = (unsigned int *)malloc(sizeof(unsigned int)*out_width);
            ProcessConflictCuda<<<n_grid,N_THREADS_PER_BLOCK>>>(_cur_level,blocks_per_inst, dev_out_datas, dev_out_times,
                _time_unit, dev_data_out_num,
                dev_out_splited_width, dev_output_start, dev_output_size, 
                dev_data_in_num_start, 
                dev_valid_width, dev_valid_width_start, dev_valid_width_size,
                dev_valid_width_flag);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                cout << "memcpy GPU to CPU kenerl error: " << hipGetErrorString(err) << endl;
            }
            //unsigned int *host_valid_width2 = (unsigned int*)malloc(sizeof(unsigned int) * length_of_valid_width);
            //err = hipMemcpy(host_valid_width2, dev_valid_width, sizeof(unsigned int)*length_of_valid_width, hipMemcpyDeviceToHost);
            

            UpdateValidWidthGPU<<<n_grid,N_THREADS_PER_BLOCK>>>(_cur_level,blocks_per_inst, dev_out_datas,
                dev_data_out_num,
                dev_out_splited_width, dev_output_start, dev_output_size, 
                dev_data_in_num_start, 
                dev_valid_width, dev_valid_width_start, dev_valid_width_size,
                dev_valid_width_flag);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                cout << "UpdateValidWidthGPU kenerl error: " << hipGetErrorString(err) << endl;
            }
            //unsigned int *host_valid_width3 = (unsigned int*)malloc(sizeof(unsigned int) * length_of_valid_width);
            //err = hipMemcpy(host_valid_width3, dev_valid_width, sizeof(unsigned int)*length_of_valid_width, hipMemcpyDeviceToHost);
            
            hipFree(dev_valid_width_flag);
            // debug
            /*if(_cur_level == 0){
            char *valid_width = new char[length_of_valid_width];
            hipMemcpy(valid_width, dev_valid_width, sizeof(char)*length_of_valid_width, hipMemcpyDeviceToHost);
            for(int c = 0; c < length_of_valid_width; ++c){
                int _c = static_cast<int>(valid_width[c]);
                cout << _c << endl;
            }
            }*/
            // Get prefix_sum of dev_valid_width, using thrust::exclusive_scan
            unsigned int *dev_prefix_sum_of_valid_width;
            err = hipMalloc((void**)&dev_prefix_sum_of_valid_width, sizeof(unsigned int) * length_of_valid_width);
            thrust::exclusive_scan(thrust::device, dev_valid_width, dev_valid_width+length_of_valid_width,  dev_prefix_sum_of_valid_width);
            // Done prefix_sum
            //unsigned int *host_prefix_sum_of_valid_width = (unsigned int*)malloc(sizeof(unsigned int) * length_of_valid_width);
            //err = hipMemcpy(host_prefix_sum_of_valid_width, dev_prefix_sum_of_valid_width, sizeof(unsigned int)*length_of_valid_width, hipMemcpyDeviceToHost);
            //if(_cur_level == 0){
            //cout << "valid_width:" << endl;
            //for(unsigned p = 0; p < length_of_valid_width; ++p){
            //    cout << host_valid_width1[p] << "," << host_valid_width2[p] << "," << host_valid_width3[p] << "--" << host_prefix_sum_of_valid_width[p] << endl;
            //}
            //}
            //free(host_valid_width1);
            //free(host_valid_width2);
            //free(host_valid_width3);
            //free(host_prefix_sum_of_valid_width);

            // Get new out_starts and out_sizes
            unsigned int *new_output_start = (unsigned int*)malloc(sizeof(unsigned int) * inst_num);
            unsigned int *new_output_size = (unsigned int*)malloc(sizeof(unsigned int) * inst_num * MAX_OUT_NUM);
            if(new_output_size == NULL){
                cout << "error malloc: new_output_size" << endl;
                exit(-1);
            }
            unsigned int *dev_out_width;
            unsigned int *dev_output_start_new;
            unsigned int *dev_output_size_new;
            
            err = hipMalloc((void **)&dev_out_width, sizeof(unsigned int));
            err = hipMalloc((void **)&dev_output_start_new, sizeof(unsigned int) * inst_num);
            err = hipMalloc((void **)&dev_output_size_new, sizeof(unsigned int) * inst_num * MAX_OUT_NUM);
            if (err != hipSuccess)
            {
                cout << "dev_output_size_new hipMalloc error: " << hipGetErrorString(err) << endl;
            }
            err = hipMemset(dev_output_size_new, 0, sizeof(unsigned int) * inst_num * MAX_OUT_NUM);
            int _block = (inst_num + UPDATE_N_THREADS_PER_BLOCK - 1) / UPDATE_N_THREADS_PER_BLOCK;
            UpdateOutInfoGPU<<<_block, UPDATE_N_THREADS_PER_BLOCK>>>(dev_prefix_sum_of_valid_width, _cur_level,inst_num,
                dev_data_out_num,
                dev_out_splited_width, dev_output_start_new, dev_output_size_new, 
                dev_data_in_num_start, 
                dev_valid_width, dev_valid_width_start, dev_valid_width_size,
                dev_out_width);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                cout << "UpdateOutInfoGPU kenerl error: " << hipGetErrorString(err) << endl;
            }
            err = hipMemcpy(&out_width, dev_out_width, sizeof(unsigned int), hipMemcpyDeviceToHost);
            err = hipMemcpy(new_output_start, dev_output_start_new, sizeof(unsigned int) * inst_num, hipMemcpyDeviceToHost);
            err = hipMemcpy(new_output_size, dev_output_size_new, sizeof(unsigned int)  * inst_num * MAX_OUT_NUM, hipMemcpyDeviceToHost);
            //cout << "new out_width:" << out_width << endl;
            //if(_cur_level == 0){
            //cout << "new_output_start/size:" << endl;
            //for(int p = 0; p < inst_num; ++p){
            //    cout << new_output_start[p] << "--" << new_output_size[p*MAX_OUT_NUM] << "," << new_output_size[p*MAX_OUT_NUM+1] << endl;
            //}
            //}
            // Remove invalid state
            unsigned int *dev_out_times_new;
            short *dev_out_datas_new;
            err = hipMalloc((void **)&dev_out_times_new, sizeof(unsigned int) * out_width);
            err = hipMalloc((void **)&dev_out_datas_new, sizeof(short) * out_width);
            RemoveInvalidStatesGPU<<<n_grid,N_THREADS_PER_BLOCK>>>(dev_out_times_new, dev_out_datas_new,
                _cur_level,blocks_per_inst, dev_out_datas, dev_out_times,
                dev_data_out_num,
                dev_out_splited_width, dev_output_start, dev_output_size, 
                dev_data_in_num_start, 
                dev_valid_width, dev_valid_width_start, dev_valid_width_size,
                dev_prefix_sum_of_valid_width);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                cout << "RemoveInvalidStatesGPU kenerl error: " << hipGetErrorString(err) << endl;
            }
            //unsigned int* host_out_times = new unsigned int[out_width];
            //short* host_out_datas = new short[out_width];
            //err = hipMemcpy(host_out_times, dev_out_times_new, sizeof(unsigned int) * out_width, hipMemcpyDeviceToHost);
            //err = hipMemcpy(host_out_datas, dev_out_datas_new, sizeof(short) * out_width, hipMemcpyDeviceToHost);
            //if(_cur_level == 0){
            //cout << "new_output:" << endl;
            //for(unsigned p = 0; p < out_width; ++p){
            //    cout << host_out_times[p] << "," << host_out_datas[p] << endl;
            //}
            //}
            //delete[] host_out_times;
            //delete[] host_out_datas;

            hipFree(dev_output_start);
            hipFree(dev_output_size);
            hipFree(dev_output_start_new);
            hipFree(dev_output_size_new);
            err=hipFree(dev_out_datas);//
            
            if (err != hipSuccess)
            {
                printf("the cudafree(dev_out_datas) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            hipFree(dev_out_times);//
            if (err != hipSuccess)
            {
                printf("the cudafree(dev_out_times) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            // Done Remove

            short *dev_out_datas_compressed;
            unsigned int dev_out_width_compressed = (out_width + STATE_NUM - 1) / STATE_NUM;
            err = hipMalloc((void **)&dev_out_datas_compressed, sizeof(short)*dev_out_width_compressed);
            CompressGPU<<<_n_grid, N_THREADS_PER_BLOCK>>>(dev_out_datas_new, dev_out_datas_compressed, out_width, dev_out_width_compressed);
            hipDeviceSynchronize();
            if (err != hipSuccess)
            {
                cout << "CompressGPU kenerl error: " << hipGetErrorString(err) << endl;
            }
            auto end_out2 = std::chrono::steady_clock::now();
            hipMemcpy(out_time, dev_out_times_new, sizeof(unsigned int)*out_width, hipMemcpyDeviceToHost);
            //hipMemcpy(out_data, dev_out_datas, sizeof(short)*out_width, hipMemcpyDeviceToHost);
            hipMemcpy(out_data, dev_out_datas_compressed, sizeof(short)*dev_out_width_compressed, hipMemcpyDeviceToHost);
            if (err != hipSuccess)
            {
                cout << "hipMemcpy dev_out_datas_compressed error: " << hipGetErrorString(err) << endl;
            }
            auto end_out = std::chrono::steady_clock::now();
            duration_out += std::chrono::duration_cast<std::chrono::milliseconds>(end_out2 - start_out).count();
            duration_out2 += std::chrono::duration_cast<std::chrono::milliseconds>(end_out - start_out).count();

            
            out_times[_cur_level] = out_time;
            out_values[_cur_level] = out_data;
            out_sizes[_cur_level] = new_output_size;
            out_starts[_cur_level] = new_output_start;
            hipFree(dev_out_datas_compressed);
            //hipFree(dev_output_size);
            //hipFree(dev_output_start);
            hipFree(dev_valid_width_start);
            hipFree(dev_valid_width_size);
            hipFree(dev_out_splited_width);//
            hipFree(dev_valid_width);//
            hipFree(dev_prefix_sum_of_valid_width);//
            hipFree(dev_out_width);//
            free(valid_width_start);
            free(valid_width_size);
            free(output_size);
            free(output_start);
            
            
            err=hipFree(dev_out_datas_new);//
            
            if (err != hipSuccess)
            {
                printf("the cudafree(dev_out_datas) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
            hipFree(dev_out_times_new);//
            
            if (err != hipSuccess)
            {
                printf("the cudafree(dev_out_datas) on GPU is failed\n");
                cout << hipGetErrorString(err) << endl;
                exit(-1);
            }
        }
        err = hipFree(dev_data_in_num);//
        checkError(err, "cudafree error");
        //hipFree(dev_val_num_start);//
        hipFree(dev_data_out_num);//
        hipFree(dev_delay_val_num);//
        hipFree(dev_functions_func_num);//
        hipFree(dev_functions_val_num);//
        hipFree(dev_delay_edges);//
        hipFree(dev_in_bit);//
        hipFree(dev_out_bit);//
        hipFree(dev_rise_val);//
        hipFree(dev_fall_val);//
        hipFree(dev_functions);//
        hipFree(dev_data_in_num_start);//
        hipFree(dev_data_out_num_start);//
        hipFree(dev_delay_start);//
        hipFree(dev_delay_width);//
        hipFree(dev_functions_start);//
        hipFree(dev_functions_width);//
        auto end = std::chrono::steady_clock::now();
        long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
        cout << "total time of simulating: " << duration << "ms" << endl;
        cout << "\ttotal time of gpu: " << duration_gpu << "ms(" << (float)((float)duration_gpu / duration) * 100.0 << "%)" << endl;
        cout << "\ttotal time of pre process: " << duration_pre << "ms(" << (float)((float)duration_pre / duration) * 100.0 << "%)" << endl;
        cout << "\t\ttime of first step: " << duration_pro0 << "ms(" << (float)((float)duration_pro0 / duration) * 100.0 << "%)" << endl;
        cout << "\t\ttime of second step: " << duration_pro00 << "us(" << (float)((float)duration_pro1 / duration) * 100.0 << "%)" << endl;
        cout << "\ttotal time of function/delay copy: " << duration_cuda2 << "ms(" << (float)((float)duration_cuda2 / duration) * 100.0 << "%)" << endl;        
        cout << "\ttotal time of data copy: " << duration_data << "ms(" << (float)((float)duration_data / duration) * 100.0 << "%)" << endl;
        cout << "\t\tmerge gpu: " << duration_malloc2 << "ms(" << (float)((float)duration_malloc2 / duration) * 100.0 << "%)" << endl;
        cout << "\t\ttotal time of data copy(not include pre process's data): " << duration_cuda << "ms(" << (float)((float)duration_cuda / duration) * 100.0 << "%)" << endl;
        cout << "\ttotal time of out process: " << duration_out2 << "ms(" << (float)((float)duration_out2 / duration) * 100.0 << "%)" << endl;
        cout << "\t\ttotal time of out process(not include out data cpy): " << duration_out << "ms(" << (float)((float)duration_out / duration) * 100.0 << "%)" << endl;
        auto start_write = std::chrono::steady_clock::now();
        //DumpSaif(outf);
        DumpSaif(outf, OutMaps, out_times, out_values, out_sizes, out_starts,
            sim_start, sim_end);
        DumpSaif(outf, parser.InMaps_bus, parser.InMaps_bit, parser.times, parser.values, parser.sizes,
            parser.vec_times, parser.vec_values, parser.vec_sizes,
            parser.hash_name_pair, parser.hash_index_pair,
            sim_start, sim_end);
        DumpSaif(outf, assign_pairs, _inter.p.Result,
            sim_start, sim_end);
        DumpSaif(outf, sim_start, sim_end);

        outf.close();
        auto end_write = std::chrono::steady_clock::now();
        long duration_write = std::chrono::duration_cast<std::chrono::milliseconds>(end_write - start).count();
        long duration_total = std::chrono::duration_cast<std::chrono::milliseconds>(end_write - start_vcd).count();
        long duration_total2 = std::chrono::duration_cast<std::chrono::milliseconds>(end_write - start_total).count();
        cout << "total time(not including read-vcd): " << duration_write << "ms" << endl;
        cout << "total time: " << duration_total << "ms" << endl;
        cout << "total time(all): " << duration_total2 << "ms" << endl;
        //DumpSaif(parser.times,parser.values, assign_pairs, out_times,out_values,out_sizes,out_starts, pinbitValues, _inter.p.pin_bits, saif_out, time_unit, time_res, root_name, sim_start, sim_end, _size1, _inter.levels, _inter);
    }
    else
    {
        std::cout << "Parse Failed." << std::endl;
        return 1;
    }
    return 0;
}